#include "hip/hip_runtime.h"
#include "..\Prerequisites.cuh"
#include "..\Functions.cuh"


////////////////////////////
//CUDA kernel declarations//
////////////////////////////

__global__ void ShiftFourierKernel(tcomplex* d_input, tcomplex* d_output, int3 dims, tfloat3 delta);
__global__ void ShiftIntegerKernel(tfloat* d_input, tfloat* d_output, int3 dims, int3 delta);


/////////////////////////////////////////////
//Equivalent of TOM's tom_cart2polar method//
/////////////////////////////////////////////

void d_Shift(tfloat* d_input, tfloat* d_output, int3 dims, tfloat3* delta, int batch)
{
	size_t elements = dims.x * dims.y * dims.z;
	size_t elementsFFT = (dims.x / 2 + 1) * dims.y * dims.z;

	tcomplex* d_intermediate;
	hipMalloc((void**)&d_intermediate, elementsFFT * sizeof(tcomplex));

	for (int b = 0; b < batch; b++)
	{
		if(fmod(delta[b].x, (tfloat)1) != (tfloat)0 || fmod(delta[b].y, (tfloat)1) != (tfloat)0 || fmod(delta[b].z, (tfloat)1) != (tfloat)0)
		{
			tfloat3 normdelta = tfloat3(delta[b].x / (tfloat)dims.x, delta[b].y / (tfloat)dims.y, delta[b].z / (tfloat)dims.z);

			d_FFTR2C(d_input + elements * b, d_intermediate, DimensionCount(dims), dims);

			int TpB = min(256, NextMultipleOf(dims.x / 2 + 1, 32));
			dim3 grid = dim3(((dims.x / 2 + 1) + TpB - 1) / TpB, dims.y, dims.z);
			ShiftFourierKernel <<<grid, TpB>>> (d_intermediate, d_intermediate, dims, normdelta);
			
			d_IFFTC2R(d_intermediate, d_output + elements * b, DimensionCount(dims), dims);
		}
		else
		{
			int TpB = min(256, NextMultipleOf(dims.x, 32));
			dim3 grid = dim3((dims.x + TpB - 1) / TpB, dims.y, dims.z);
			ShiftIntegerKernel <<<grid, TpB>>> (d_input + elements * b, d_input == d_output ? (tfloat*)d_intermediate : (d_output + elements * b), dims, toInt3((int)delta[b].x, (int)delta[b].y, (int)delta[b].z));

			if(d_input == d_output)
				hipMemcpy(d_output + elements * b, d_intermediate, elements * sizeof(tfloat), hipMemcpyDeviceToDevice);
		}
	}

	hipFree(d_intermediate);
}


////////////////
//CUDA kernels//
////////////////

__global__ void ShiftFourierKernel(tcomplex* d_input, tcomplex* d_output, int3 dims, tfloat3 delta)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	if(x >= dims.x / 2 + 1)
		return;
	if(x == dims.x / 2)
		x = (-x);
	int y = ((blockIdx.y + ((dims.y + 1) / 2)) % dims.y) - (dims.y / 2);
	int z = ((blockIdx.z + ((dims.z + 1) / 2)) % dims.z) - (dims.z / 2);

	tfloat factorx = delta.x * (tfloat)x * (tfloat)PI2;
	tcomplex multx = { cos(factorx), sin(-factorx) };
	if(blockIdx.x * blockDim.x + threadIdx.x == dims.x / 2)
		multx.y = (tfloat)0;
	tfloat factory = delta.y * (tfloat)y * (tfloat)PI2;
	tcomplex multy = { cos(factory), sin(-factory) };
	if(blockIdx.y == dims.y / 2)
		multy.y = (tfloat)0;

	tcomplex multiplicator = cmul(multx, multy);

	if(dims.z > 1)
	{
		tfloat factorz = delta.z * (tfloat)z * (tfloat)PI2;
		tcomplex multz = { cos(factorz), sin(-factorz) };
		if(blockIdx.z == dims.z / 2)
			multz.y = (tfloat)0;
		multiplicator = cmul(multiplicator, multz);
	}	

	size_t id = (blockIdx.z * dims.y + blockIdx.y) * (dims.x / 2 + 1) + blockIdx.x * blockDim.x + threadIdx.x;
	d_output[id] = cmul(d_input[id], multiplicator);
	//d_output[id] = multiplicator;
}

__global__ void ShiftIntegerKernel(tfloat* d_input, tfloat* d_output, int3 dims, int3 delta)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;

	d_output[((abs(dims.z + (int)blockIdx.z) % dims.z) * dims.y + (abs(dims.y + (int)blockIdx.y) % dims.y)) * dims.x + (abs(dims.x + x) % dims.x)] = d_input[(blockIdx.z * dims.y + blockIdx.y) * dims.x + x];
}