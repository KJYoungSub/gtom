#include "hip/hip_runtime.h"
#include "..\Prerequisites.cuh"
#include "..\Functions.cuh"


////////////////////////////
//CUDA kernel declarations//
////////////////////////////

__global__ void ShiftFourierKernel(tcomplex* d_input, tcomplex* d_output, int3 dims, tfloat3 delta);
__global__ void ShiftIntegerKernel(tfloat* d_input, tfloat* d_output, int3 dims, int3 delta);


/////////////////////////////////////////////
//Equivalent of TOM's tom_cart2polar method//
/////////////////////////////////////////////

void d_Shift(tfloat* d_input, tfloat* d_output, int3 dims, tfloat3* delta, int batch)
{
	tcomplex* d_intermediate;
	hipMalloc((void**)&d_intermediate, ElementsFFT(dims) * sizeof(tcomplex));

	for (int b = 0; b < batch; b++)
	{
		if(fmod(delta[b].x, (tfloat)1) != (tfloat)0 || fmod(delta[b].y, (tfloat)1) != (tfloat)0 || fmod(delta[b].z, (tfloat)1) != (tfloat)0)
		{
			tfloat3 normdelta = tfloat3(delta[b].x / (tfloat)dims.x, delta[b].y / (tfloat)dims.y, delta[b].z / (tfloat)dims.z);

			d_FFTR2C(d_input + Elements(dims) * b, d_intermediate, DimensionCount(dims), dims);

			int TpB = min(256, NextMultipleOf(dims.x / 2 + 1, 32));
			dim3 grid = dim3(((dims.x / 2 + 1) + TpB - 1) / TpB, dims.y, dims.z);
			ShiftFourierKernel <<<grid, TpB>>> (d_intermediate, d_intermediate, dims, normdelta);
			
			d_IFFTC2R(d_intermediate, d_output + Elements(dims) * b, DimensionCount(dims), dims);
		}
		else
		{
			int TpB = min(256, NextMultipleOf(dims.x, 32));
			dim3 grid = dim3((dims.x + TpB - 1) / TpB, dims.y, dims.z);
			ShiftIntegerKernel <<<grid, TpB>>> (d_input + Elements(dims) * b, d_input == d_output ? (tfloat*)d_intermediate : (d_output + Elements(dims) * b), dims, toInt3((int)delta[b].x, (int)delta[b].y, (int)delta[b].z));

			if(d_input == d_output)
				hipMemcpy(d_output + Elements(dims) * b, d_intermediate, Elements(dims) * sizeof(tfloat), hipMemcpyDeviceToDevice);
		}
	}

	hipFree(d_intermediate);
}


////////////////
//CUDA kernels//
////////////////

__global__ void ShiftFourierKernel(tcomplex* d_input, tcomplex* d_output, int3 dims, tfloat3 delta)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	if(x >= dims.x / 2 + 1)
		return;
	if(x == dims.x / 2)
		x = (-x);
	int y = ((blockIdx.y + ((dims.y + 1) / 2)) % dims.y) - (dims.y / 2);
	int z = ((blockIdx.z + ((dims.z + 1) / 2)) % dims.z) - (dims.z / 2);

	tfloat factorx = delta.x * (tfloat)x * (tfloat)PI2;
	tcomplex multx = { cos(factorx), sin(-factorx) };
	if(dims.x % 2 == 0 && blockIdx.x * blockDim.x + threadIdx.x == dims.x / 2)
		multx.y = (tfloat)0;

	tfloat factory = delta.y * (tfloat)y * (tfloat)PI2;
	tcomplex multy = { cos(factory), sin(-factory) };
	if(dims.y % 2 == 0 && blockIdx.y == dims.y / 2)
		multy.y = (tfloat)0;

	tcomplex multiplicator = cmul(multx, multy);

	if(dims.z > 1)
	{
		tfloat factorz = delta.z * (tfloat)z * (tfloat)PI2;
		tcomplex multz = { cos(factorz), sin(-factorz) };
		if(dims.z % 2 == 0 && blockIdx.z == dims.z / 2)
			multz.y = (tfloat)0;
		multiplicator = cmul(multiplicator, multz);
	}	

	size_t id = (blockIdx.z * dims.y + blockIdx.y) * (dims.x / 2 + 1) + blockIdx.x * blockDim.x + threadIdx.x;
	d_output[id] = cmul(d_input[id], multiplicator);
	//d_output[id] = multiplicator;
}

__global__ void ShiftIntegerKernel(tfloat* d_input, tfloat* d_output, int3 dims, int3 delta)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	if(x >= dims.x)
		return;

	d_output[(((dims.z + (int)blockIdx.z + delta.z) % dims.z) * dims.y + ((dims.y + (int)blockIdx.y + delta.y) % dims.y)) * dims.x + ((dims.x + x + delta.x) % dims.x)] = d_input[(blockIdx.z * dims.y + blockIdx.y) * dims.x + x];
}