#include "hip/hip_runtime.h"
#include "../Prerequisites.cuh"
#include "../Functions.cuh"

////////////////////////////
//CUDA kernel declarations//
////////////////////////////

__global__ void FFTCropEvenKernel(tcomplex* d_input, tcomplex* d_output, int3 olddims, int3 newdims);
//__global__ void FFTCropOddKernel(tcomplex* d_input, tcomplex* d_output, int3 olddims, int3 newdims);
__global__ void FFTPadEvenKernel(tcomplex* d_input, tcomplex* d_output, int3 olddims, int3 newdims);

__host__ __device__ tcomplex toTcomplex(tfloat r, tfloat i)
{
	tcomplex value = {r, i};
	return value;
}


////////////////
//Host methods//
////////////////

void d_FFTCrop(tcomplex* d_input, tcomplex* d_output, int3 olddims, int3 newdims, int batch)
{
	size_t elementsnew = (newdims.x / 2 + 1) * newdims.y * newdims.z;
	size_t elementsold = (olddims.x / 2 + 1) * olddims.y * olddims.z;

	int TpB = min(256, NextMultipleOf(newdims.x / 2 + 1, 32));
	dim3 grid = dim3((newdims.x + TpB - 1) / TpB, newdims.y, newdims.z);
	if(newdims.x % 2 == 0)
		for(int b = 0; b < batch; b++)
			FFTCropEvenKernel <<<grid, TpB>>> (d_input + elementsold * b, d_output + elementsnew * b, olddims, newdims);
	/*else
		for(int b = 0; b < batch; b++)
			FFTCropOddKernel <<<grid, TpB>>> (d_input + elementsold * b, d_output + elementsnew * b, olddims, newdims);*/

	hipDeviceSynchronize();
}

void d_FFTPad(tcomplex* d_input, tcomplex* d_output, int3 olddims, int3 newdims, int batch)
{
	size_t elementsnew = (newdims.x / 2 + 1) * newdims.y * newdims.z;
	size_t elementsold = (olddims.x / 2 + 1) * olddims.y * olddims.z;

	int TpB = min(256, NextMultipleOf(newdims.x / 2 + 1, 32));
	dim3 grid = dim3((newdims.x + TpB - 1) / TpB, newdims.y, newdims.z);
	//if(newdims.x % 2 == 0)
		for(int b = 0; b < batch; b++)
			FFTPadEvenKernel <<<grid, TpB>>> (d_input + elementsold * b, d_output + elementsnew * b, olddims, newdims);
	//else
		//for(int b = 0; b < batch; b++)
			//FFTCropOddKernel <<<grid, TpB>>> (d_input + elementsold * b, d_output + elementsnew * b, olddims, newdims);

	hipDeviceSynchronize();
}


////////////////
//CUDA kernels//
////////////////

__global__ void FFTCropEvenKernel(tcomplex* d_input, tcomplex* d_output, int3 olddims, int3 newdims)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	if(x >= newdims.x / 2 + 1)
		return;

	int newry = ((blockIdx.y + newdims.y / 2) % newdims.y);
	int newrz = ((blockIdx.z + newdims.z / 2) % newdims.z);

	int oldry = (olddims.y - newdims.y) / 2 + newry;
	int oldrz = (olddims.z - newdims.z) / 2 + newrz;

	int oldy = ((oldry + (olddims.y + 1) / 2) % olddims.y);
	int oldz = ((oldrz + (olddims.z + 1) / 2) % olddims.z);

	if(x == newdims.x / 2)
	{
		if(oldy != 0)
			oldy = olddims.y - oldy;
		if(oldz != 0)
			oldz = olddims.z - oldz;
	}

	tcomplex val = d_input[(oldz * olddims.y + oldy) * (olddims.x / 2 + 1) + x];
	if(x == newdims.x / 2)
		d_output[(blockIdx.z * newdims.y + blockIdx.y) * (newdims.x / 2 + 1) + x] = cconj(val);
	else
		d_output[(blockIdx.z * newdims.y + blockIdx.y) * (newdims.x / 2 + 1) + x] = val;
}

//__global__ void FFTCropOddKernel(tcomplex* d_input, tcomplex* d_output, int3 olddims, int3 newdims)
//{
//	int x = blockIdx.x * blockDim.x + threadIdx.x;
//	if(x >= newdims.x / 2 + 1)
//		return;
//
//	int newry = ((blockIdx.y + newdims.y / 2) % newdims.y);
//	int newrz = ((blockIdx.z + newdims.z / 2) % newdims.z);
//
//	int oldry = (olddims.y - newdims.y) / 2 + newry;
//	int oldrz = (olddims.z - newdims.z) / 2 + newrz;
//
//	int oldy = ((oldry + (olddims.y + 1) / 2) % olddims.y);
//	int oldz = ((oldrz + (olddims.z + 1) / 2) % olddims.z);
//
//	d_output[(blockIdx.z * newdims.y + blockIdx.y) * (newdims.x / 2 + 1) + x] = d_input[(oldz * olddims.y + oldy) * (olddims.x / 2 + 1) + x];
//}

__global__ void FFTPadEvenKernel(tcomplex* d_input, tcomplex* d_output, int3 olddims, int3 newdims)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	if(x >= newdims.x / 2 + 1)
		return;

	int newry = ((blockIdx.y + newdims.y / 2) % newdims.y);
	int newrz = ((blockIdx.z + newdims.z / 2) % newdims.z);

	int oldry =  newry + (olddims.y - newdims.y) / 2;
	int oldrz =  newrz + (olddims.z - newdims.z) / 2;
	
	if(x < olddims.x / 2 && oldry >= 0 && oldry < olddims.y && oldrz >= 0 && oldrz < olddims.z)
	{
		int oldy = ((oldry + (olddims.y + 1) / 2) % olddims.y);
		int oldz = ((oldrz + (olddims.z + 1) / 2) % olddims.z);

		d_output[(blockIdx.z * newdims.y + blockIdx.y) * (newdims.x / 2 + 1) + x] = d_input[(oldz * olddims.y + oldy) * (olddims.x / 2 + 1) + x];
	}
	else
		d_output[(blockIdx.z * newdims.y + blockIdx.y) * (newdims.x / 2 + 1) + x] = toTcomplex((tfloat)0, (tfloat)0);
	/*if(x == newdims.x / 2)
	{
		if(oldy != 0)
			oldy = olddims.y - oldy;
		if(oldz != 0)
			oldz = olddims.z - oldz;
	}*/

}