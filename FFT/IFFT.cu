#include "hip/hip_runtime.h"
#include "..\Prerequisites.cuh"
#include "..\Functions.cuh"

void d_IFFTC2R(tcomplex* const d_input, tfloat* const d_output, int const ndimensions, int3 const dimensions)
{
	hipfftHandle plan;
	hipfftType direction = IS_TFLOAT_DOUBLE ? HIPFFT_Z2D : HIPFFT_C2R;

	if(ndimensions == 1)
		CudaSafeCall((hipError_t)hipfftPlan1d(&plan, dimensions.x, direction, 1));
	else if(ndimensions == 2)
		CudaSafeCall((hipError_t)hipfftPlan2d(&plan, dimensions.y, dimensions.x, direction));
	else if(ndimensions == 3)
		CudaSafeCall((hipError_t)hipfftPlan3d(&plan, dimensions.z, dimensions.y, dimensions.x, direction));
	else
		throw;

	CudaSafeCall((hipError_t)cufftSetCompatibilityMode(plan, CUFFT_COMPATIBILITY_NATIVE));
	#ifdef TOM_DOUBLE
		CUDA_MEASURE_TIME(CudaSafeCall((hipError_t)hipfftExecZ2D(plan, d_input, d_output)));
	#else
		CUDA_MEASURE_TIME(CudaSafeCall((hipError_t)hipfftExecC2R(plan, d_input, d_output)));
	#endif

	CudaSafeCall((hipError_t)hipfftDestroy(plan));

	size_t elements = dimensions.x * dimensions.y * dimensions.z;
	d_Multiply(d_output, d_output, elements, 1.0f / (float)elements);
}

void d_IFFTC2RFull(tcomplex* const d_input, tfloat* const d_output, int const ndimensions, int3 const dimensions)
{
	tcomplex* d_unpadded;
	hipMalloc((void**)&d_unpadded, (dimensions.x / 2 + 1) * dimensions.y * dimensions.z * sizeof(tcomplex));

	d_HermitianSymmetryTrim(d_input, d_unpadded, dimensions);
	hipDeviceSynchronize();
	d_IFFTC2R(d_unpadded, d_output, ndimensions, dimensions);

	hipFree(d_unpadded);
}

void d_IFFTC2C(tcomplex* const d_input, tcomplex* const d_output, int const ndimensions, int3 const dimensions)
{
	hipfftHandle plan;
	hipfftType direction = IS_TFLOAT_DOUBLE ? HIPFFT_Z2Z : HIPFFT_C2C;

	if(ndimensions == 1)
		CudaSafeCall((hipError_t)hipfftPlan1d(&plan, dimensions.x, direction, 1));
	else if(ndimensions == 2)
		CudaSafeCall((hipError_t)hipfftPlan2d(&plan, dimensions.y, dimensions.x, direction));
	else if(ndimensions == 3)
		CudaSafeCall((hipError_t)hipfftPlan3d(&plan, dimensions.z, dimensions.y, dimensions.x, direction));
	else
		throw;

	CudaSafeCall((hipError_t)cufftSetCompatibilityMode(plan, CUFFT_COMPATIBILITY_NATIVE));
	#ifdef TOM_DOUBLE
		CUDA_MEASURE_TIME(CudaSafeCall((hipError_t)hipfftExecZ2Z(plan, d_input, d_output)));
	#else
		CUDA_MEASURE_TIME(CudaSafeCall((hipError_t)hipfftExecC2C(plan, d_input, d_output, HIPFFT_BACKWARD)));
	#endif
	
	CudaSafeCall((hipError_t)hipfftDestroy(plan));

	size_t elements = dimensions.x * dimensions.y * dimensions.z ;
	d_Multiply((tfloat*)d_output, (tfloat*)d_output, elements * 2, 1.0f / (float)elements);
}

void IFFTC2R(tcomplex* const h_input, tfloat* const h_output, int const ndimensions, int3 const dimensions)
{
	size_t reallength = dimensions.x * dimensions.y * dimensions.z;
	size_t complexlength = (dimensions.x / 2 + 1) * dimensions.y * dimensions.z;

	tcomplex* d_A = (tcomplex*)CudaMallocFromHostArray(h_input, complexlength * sizeof(tcomplex));

	d_IFFTC2R(d_A, (tfloat*)d_A, ndimensions, dimensions);

	hipMemcpy(h_output, d_A, reallength * sizeof(tfloat), hipMemcpyDeviceToHost);
	hipFree(d_A);
}

void IFFTC2RFull(tcomplex* const h_input, tfloat* const h_output, int const ndimensions, int3 const dimensions)
{
	size_t reallength = dimensions.x * dimensions.y * dimensions.z;
	size_t complexlength = dimensions.x * dimensions.y * dimensions.z;

	tcomplex* d_A = (tcomplex*)CudaMallocFromHostArray(h_input, complexlength * sizeof(tcomplex));
	//tfloat* d_B;
	//hipMalloc((void**)&d_B, reallength * sizeof(tfloat));

	d_IFFTC2RFull(d_A, (tfloat*)d_A, ndimensions, dimensions);

	hipMemcpy(h_output, d_A, reallength * sizeof(tfloat), hipMemcpyDeviceToHost);
	hipFree(d_A);
	//hipFree(d_B);
}

void IFFTC2C(tcomplex* const h_input, tcomplex* const h_output, int const ndimensions, int3 const dimensions)
{
	size_t complexlength = dimensions.x * dimensions.y * dimensions.z;

	tcomplex* d_A = (tcomplex*)CudaMallocFromHostArray(h_input, complexlength * sizeof(tcomplex));

	d_IFFTC2C(d_A, d_A, ndimensions, dimensions);

	hipMemcpy(h_output, d_A, complexlength * sizeof(tcomplex), hipMemcpyDeviceToHost);
	hipFree(d_A);
}