#include "hip/hip_runtime.h"
#include "..\Prerequisites.cuh"
#include "..\Functions.cuh"

__declspec(dllexport) void __stdcall FFT(tfloat* const d_input, tcomplex* const d_output, int const ndimensions, int3 const dimensions)
{
	hipfftHandle plan;
	#ifdef TOM_DOUBLE
		#define HIPFFT_FORWARD HIPFFT_D2Z
	#else
		#define HIPFFT_FORWARD HIPFFT_R2C
	#endif

	switch (ndimensions)
	{
		case 1:
			CudaSafeCall((hipError_t)hipfftPlan1d(&plan, dimensions.x, HIPFFT_FORWARD, 1));
			break;
		case 2:
			CudaSafeCall((hipError_t)hipfftPlan2d(&plan, dimensions.x, dimensions.y, HIPFFT_FORWARD));
			break;
		case 3:
			CudaSafeCall((hipError_t)hipfftPlan3d(&plan, dimensions.x, dimensions.y, dimensions.z, HIPFFT_FORWARD));
			break;
		default:
			throw;
			break;
	}

	CudaSafeCall((hipError_t)cufftSetCompatibilityMode(plan, CUFFT_COMPATIBILITY_NATIVE));
	#ifdef TOM_DOUBLE
		CudaSafeCall((hipError_t)hipfftExecD2Z(plan, d_input, d_output));
	#else
		CudaSafeCall((hipError_t)hipfftExecR2C(plan, d_input, d_output));
	#endif
	
	CudaSafeCall((hipError_t)hipfftDestroy(plan));
}