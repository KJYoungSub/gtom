#include "hip/hip_runtime.h"
#include "..\Prerequisites.cuh"
#include "..\Functions.cuh"

__declspec(dllexport) void __stdcall FFT(hipfftReal* const d_input, hipfftComplex* const d_output, int const ndimensions, int3 const dimensions)
{
	hipfftHandle plan;

	switch (ndimensions)
	{
		case 1:
			CudaSafeCall((hipError_t)hipfftPlan1d(&plan, dimensions.x, HIPFFT_R2C, 1));
			break;
		case 2:
			CudaSafeCall((hipError_t)hipfftPlan2d(&plan, dimensions.x, dimensions.y, HIPFFT_R2C));
			break;
		case 3:
			CudaSafeCall((hipError_t)hipfftPlan3d(&plan, dimensions.x, dimensions.y, dimensions.z, HIPFFT_R2C));
			break;
		default:
			throw;
			break;
	}

	CudaSafeCall((hipError_t)cufftSetCompatibilityMode(plan, CUFFT_COMPATIBILITY_NATIVE));
	CudaSafeCall((hipError_t)hipfftExecR2C(plan, d_input, d_output));
	
	CudaSafeCall((hipError_t)hipfftDestroy(plan));
}