#include "hip/hip_runtime.h"
#include "..\Prerequisites.cuh"
#include "..\Functions.cuh"

void d_FFTR2C(tfloat* const d_input, tcomplex* const d_output, int const ndimensions, int3 const dimensions, int batch)
{
	hipfftHandle plan;
	hipfftType direction = IS_TFLOAT_DOUBLE ? HIPFFT_D2Z : HIPFFT_R2C;
	int n[3] = { dimensions.z, dimensions.y, dimensions.x };

	CudaSafeCall((hipError_t)hipfftPlanMany(&plan, ndimensions, n + (3 - ndimensions),
										  NULL, 1, 0,
										  NULL, 1, 0,
										  direction, batch));

	CudaSafeCall((hipError_t)cufftSetCompatibilityMode(plan, CUFFT_COMPATIBILITY_NATIVE));
	#ifdef TOM_DOUBLE
		CudaSafeCall((hipError_t)hipfftExecD2Z(plan, d_input, d_output));
	#else
		CudaSafeCall((hipError_t)hipfftExecR2C(plan, d_input, d_output));
	#endif
	
	hipDeviceSynchronize();
	
	CudaSafeCall((hipError_t)hipfftDestroy(plan));
}

void d_FFTR2CFull(tfloat* const d_input, tcomplex* const d_output, int const ndimensions, int3 const dimensions, int batch)
{
	tcomplex* d_unpadded;
	hipMalloc((void**)&d_unpadded, (dimensions.x / 2 + 1) * dimensions.y * dimensions.z * batch * sizeof(tcomplex));

	d_FFTR2C(d_input, d_unpadded, ndimensions, dimensions, batch);
	d_HermitianSymmetryPad(d_unpadded, d_output, dimensions, batch);

	hipFree(d_unpadded);
}

void d_FFTC2C(tcomplex* const d_input, tcomplex* const d_output, int const ndimensions, int3 const dimensions, int batch)
{
	hipfftHandle plan;
	hipfftType direction = IS_TFLOAT_DOUBLE ? HIPFFT_Z2Z : HIPFFT_C2C;
	int n[3] = { dimensions.z, dimensions.y, dimensions.x };

	CudaSafeCall((hipError_t)hipfftPlanMany(&plan, ndimensions, n + (3 - ndimensions),
										  NULL, 1, 0,
										  NULL, 1, 0,
										  direction, batch));

	CudaSafeCall((hipError_t)cufftSetCompatibilityMode(plan, CUFFT_COMPATIBILITY_NATIVE));
	#ifdef TOM_DOUBLE
		CudaSafeCall((hipError_t)hipfftExecZ2Z(plan, d_input, d_output));
	#else
		CudaSafeCall((hipError_t)hipfftExecC2C(plan, d_input, d_output, HIPFFT_FORWARD));
	#endif
	
	CudaSafeCall((hipError_t)hipfftDestroy(plan));
}

void FFTR2C(tfloat* const h_input, tcomplex* const h_output, int const ndimensions, int3 const dimensions, int batch)
{
	size_t reallength = dimensions.x * dimensions.y * dimensions.z;
	size_t complexlength = (dimensions.x / 2 + 1) * dimensions.y * dimensions.z;

	tfloat* d_A;
	hipMalloc((void**)&d_A, complexlength * batch * sizeof(tcomplex));
	for (int b = 0; b < batch; b++)
		hipMemcpy(d_A + complexlength * 2 * b, h_input + reallength * b, reallength * sizeof(tfloat), hipMemcpyHostToDevice);

	d_FFTR2C(d_A, (tcomplex*)d_A, ndimensions, dimensions, batch);

	hipMemcpy(h_output, d_A, complexlength * batch * sizeof(tcomplex), hipMemcpyDeviceToHost);
	hipFree(d_A);
}

void FFTR2CFull(tfloat* const h_input, tcomplex* const h_output, int const ndimensions, int3 const dimensions, int batch)
{
	size_t reallength = dimensions.x * dimensions.y * dimensions.z;
	size_t complexlength = dimensions.x * dimensions.y * dimensions.z;

	tfloat* d_A;
	hipMalloc((void**)&d_A, complexlength * batch * sizeof(tcomplex));
	for (int b = 0; b < batch; b++)
		hipMemcpy(d_A + complexlength * 2 * b, h_input + reallength * b, reallength * sizeof(tfloat), hipMemcpyHostToDevice);

	d_FFTR2CFull(d_A, (tcomplex*)d_A, ndimensions, dimensions, batch);

	hipMemcpy(h_output, d_A, reallength * batch * sizeof(tcomplex), hipMemcpyDeviceToHost);
	hipFree(d_A);
}

void FFTC2C(tcomplex* const h_input, tcomplex* const h_output, int const ndimensions, int3 const dimensions, int batch)
{
	size_t complexlength = dimensions.x * dimensions.y * dimensions.z;

	tcomplex* d_A = (tcomplex*)CudaMallocFromHostArray(h_input, complexlength * batch * sizeof(tcomplex));

	d_FFTC2C(d_A, d_A, ndimensions, dimensions, batch);

	hipMemcpy(h_output, d_A, complexlength * batch * sizeof(tcomplex), hipMemcpyDeviceToHost);
	hipFree(d_A);
}