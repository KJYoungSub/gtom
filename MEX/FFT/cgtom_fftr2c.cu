#include "hip/hip_runtime.h"
#include "..\Prerequisites.h"


void mexFunction(int nlhs, mxArray *plhs[],
                 int nrhs, mxArray const *prhs[])
{
    char const * const errId = "parallel:gpu:mexGPUExample:InvalidInput";
    char const * const errMsg = "Invalid input to MEX file.";

    mxInitGPU();

	if (nrhs != 1 || mxIsComplex(prhs[0]))
        mexErrMsgIdAndTxt(errId, errMsg);

	mxArrayAdapter A(prhs[0]);
	int ndims = mxGetNumberOfDimensions(A.underlyingarray);
	if (ndims < 1 || ndims > 3)
		mexErrMsgIdAndTxt(errId, errMsg);
	int3 dimensions = MWDimsToInt3(ndims, mxGetDimensions(A.underlyingarray));
	tcomplex* d_result;
	hipMalloc((void**)&d_result, (dimensions.x / 2 + 1) * dimensions.y * dimensions.z * sizeof(tcomplex));

    d_FFTR2C(A.GetAsManagedDeviceTFloat(), d_result, ndims, dimensions);

	mwSize complexdims[3] = { dimensions.x / 2 + 1, dimensions.y, dimensions.z };
	mxArrayAdapter B(mxCreateNumericArray(mxGetNumberOfDimensions(A.underlyingarray),
					 complexdims,
					 mxGetClassID(A.underlyingarray),
					 mxCOMPLEX));
	B.SetFromDeviceTComplex(d_result);
	plhs[0] = B.underlyingarray;

	hipFree(d_result);
}