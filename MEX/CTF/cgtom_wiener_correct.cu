#include "hip/hip_runtime.h"
#include "..\Prerequisites.h"


void mexFunction(int nlhs, mxArray *plhs[],
	int nrhs, mxArray const *prhs[])
{
	char const * const errId = "GTOM:CTF:WienerCorrect:InvalidInput";

	mxInitGPU();

	if (nrhs != 3)
		mexErrMsgIdAndTxt(errId, "Wrong parameter count (3 expected).");

	mxArrayAdapter image(prhs[0]);
	int3 dimsimage = MWDimsToInt3(mxGetNumberOfDimensions(image.underlyingarray), mxGetDimensions(image.underlyingarray));
	tfloat* d_image = image.GetAsManagedDeviceTFloat();
	int nimages = dimsimage.z;
	dimsimage.z = 1;

	mxArrayAdapter params(prhs[1]);
	int3 dimsparams = MWDimsToInt3(mxGetNumberOfDimensions(params.underlyingarray), mxGetDimensions(params.underlyingarray));
	if (dimsparams.x % 11 != 0)
		mexErrMsgIdAndTxt(errId, "CTF parameters should have 11 elements per image.");
	CTFParams* h_params = (CTFParams*)params.GetAsManagedTFloat();

	mxArrayAdapter fsc(prhs[2]);
	int3 dimsfsc = MWDimsToInt3(mxGetNumberOfDimensions(fsc.underlyingarray), mxGetDimensions(fsc.underlyingarray));
	if (dimsfsc.x != dimsimage.x / 2 || dimsfsc.y != nimages)
		mexErrMsgIdAndTxt(errId, "FSC curve matrix should have [size(image, 1) / 2] elements in first, and [size(image, 3)] elements in second dimension.");
	tfloat* d_fsc = fsc.GetAsManagedDeviceTFloat();

	tcomplex* d_imageft;
	hipMalloc((void**)&d_imageft, ElementsFFT(dimsimage) * nimages * sizeof(tcomplex));
	tfloat* d_weights;
	hipMalloc((void**)&d_weights, ElementsFFT(dimsimage) * nimages * sizeof(tfloat));

	d_FFTR2C(d_image, d_imageft, 2, dimsimage, nimages);

	for (int n = 0; n < nimages; n++)
		d_WienerPerFreq(d_imageft + ElementsFFT(dimsimage) * n, dimsimage, d_fsc + (dimsimage.x / 2) * n, h_params[n], d_imageft + ElementsFFT(dimsimage) * n, d_weights + ElementsFFT(dimsimage) * n);

	d_IFFTC2R(d_imageft, d_image, 2, dimsimage, nimages);

	mwSize outputdims[3];
	outputdims[0] = dimsimage.x;
	outputdims[1] = dimsimage.y;
	outputdims[2] = nimages;
	mxArrayAdapter A(mxCreateNumericArray(3,
		outputdims,
		mxGetClassID(image.underlyingarray),
		mxREAL));
	A.SetFromDeviceTFloat(d_image);
	plhs[0] = A.underlyingarray;

	mwSize fitdims[3];
	fitdims[0] = dimsimage.x / 2 + 1;
	fitdims[1] = dimsimage.y;
	fitdims[2] = nimages;
	mxArrayAdapter B(mxCreateNumericArray(3,
		fitdims,
		mxSINGLE_CLASS,
		mxREAL));
	B.SetFromDeviceTFloat(d_weights);
	plhs[1] = B.underlyingarray;

	hipFree(d_imageft);
	hipFree(d_weights);
}