#include "hip/hip_runtime.h"
#include "../Prerequisites.cuh"
#include "../Functions.cuh"
#include "../GLMFunctions.cuh"

#define GLM_FORCE_RADIANS
#define GLM_FORCE_INLINE
#define GLM_FORCE_CUDA
#include "../glm/glm.hpp"
#include "../glm/gtc/matrix_transform.hpp"
#include "../glm/gtx/quaternion.hpp"
#include "../glm/gtx/euler_angles.hpp"
#include "../glm/gtc/type_ptr.hpp"


////////////////////////////
//CUDA kernel declarations//
////////////////////////////

__global__ void ReconstructFourierKernel(tcomplex* d_projft, tcomplex* d_volumeft, tfloat* d_samples, int3 dimsvolume, int3 dimsproj, glm::vec3* d_vecX, glm::vec3* d_vecY);

////////////////////////////////////////
//Performs 3D reconstruction using ART//
////////////////////////////////////////

void d_ReconstructFourier(tfloat* d_projections, int3 dimsproj, tfloat* d_volume, int3 dimsvolume, tfloat2* h_angles)
{
	d_RemapFull2FullFFT(d_projections, d_projections, toInt3(dimsproj.x, dimsproj.y, 1), dimsproj.z);

	tcomplex* d_projft;
	hipMalloc((void**)&d_projft, ElementsFFT(dimsproj) * sizeof(tcomplex));
	for (int b = 0; b < dimsproj.z; b++)
		d_FFTR2C(d_projections + dimsproj.x * dimsproj.y * b, d_projft + (dimsproj.x / 2 + 1) * dimsproj.y * b, 2, toInt3(dimsproj.x, dimsproj.y, 1));

	tcomplex* d_projftsym;
	hipMalloc((void**)&d_projftsym, Elements(dimsproj) * sizeof(tcomplex));
	d_HermitianSymmetryPad(d_projft, d_projftsym, toInt3(dimsproj.x, dimsproj.y, 1), dimsproj.z);

	tcomplex* d_projftshifted;
	//hipMalloc((void**)&d_projftshifted, ElementsFFT(dimsproj) * sizeof(tcomplex));
	//d_RemapHalfFFT2Half(d_projft, d_projftshifted, toInt3(dimsproj.x, dimsproj.y, 1), dimsproj.z);
	hipMalloc((void**)&d_projftshifted, Elements(dimsproj) * sizeof(tcomplex));
	d_RemapFullFFT2Full(d_projftsym, d_projftshifted, toInt3(dimsproj.x, dimsproj.y, 1), dimsproj.z);
	hipFree(d_projft);
	hipFree(d_projftsym);

	//tcomplex* h_projftshifted = (tcomplex*)MallocFromDeviceArray(d_projftshifted, ElementsFFT(dimsproj) * sizeof(tcomplex));
	//free(h_projftshifted);

	//tcomplex* d_volumeft = (tcomplex*)CudaMallocValueFilled(ElementsFFT(dimsvolume) * 2, (tfloat)0);
	//tfloat* d_samples = CudaMallocValueFilled(ElementsFFT(dimsvolume), (tfloat)0);
	tcomplex* d_volumeft = (tcomplex*)CudaMallocValueFilled(Elements(dimsvolume) * 2, (tfloat)0);
	tfloat* d_samples = CudaMallocValueFilled(Elements(dimsvolume), (tfloat)0);

	glm::vec3* h_vecX = (glm::vec3*)malloc(dimsproj.z * sizeof(glm::vec3));
	glm::vec3* h_vecY = (glm::vec3*)malloc(dimsproj.z * sizeof(glm::vec3));

	glm::vec4 vecX(1.0f, 0.0f, 0.0f, 1.0f);
	glm::vec4 vecY(0.0f, 1.0f, 0.0f, 1.0f);

	for (int b = 0; b < dimsproj.z; b++)
	{
		glm::mat4 rotationmat = GetEulerRotation(h_angles[b]);
		glm::vec4 transvecX = vecX * rotationmat;
		h_vecX[b] = glm::vec3(transvecX.x, transvecX.y, transvecX.z);
		glm::vec4 transvecY = vecY * rotationmat;
		h_vecY[b] = glm::vec3(transvecY.x, transvecY.y, transvecY.z);
	}

	glm::vec3* d_vecX = (glm::vec3*)CudaMallocFromHostArray(h_vecX, dimsproj.z * sizeof(glm::vec3));
	glm::vec3* d_vecY = (glm::vec3*)CudaMallocFromHostArray(h_vecY, dimsproj.z * sizeof(glm::vec3));
	
	free(h_vecX);
	free(h_vecY);

	int TpB = min(NextMultipleOf((dimsproj.x) * dimsproj.y, 32), 256);
	dim3 grid = dim3(((dimsproj.x) * dimsproj.y + TpB - 1) / TpB, dimsproj.z);
	ReconstructFourierKernel <<<grid, TpB>>> (d_projftshifted, d_volumeft, d_samples, dimsvolume, toInt3(dimsproj.x, dimsproj.y, 1), d_vecX, d_vecY);

	hipFree(d_vecX);
	hipFree(d_vecY);
	hipFree(d_projftshifted);

	d_Inv(d_samples, d_samples, Elements(dimsvolume));
	d_ComplexMultiplyByVector(d_volumeft, d_samples, d_volumeft, Elements(dimsvolume));
	hipFree(d_samples);
	d_RemapFull2FullFFT(d_volumeft, d_volumeft, dimsvolume);

	tcomplex* d_volumeftasym;
	hipMalloc((void**)&d_volumeftasym, ElementsFFT(dimsvolume) * sizeof(tcomplex));
	d_HermitianSymmetryTrim(d_volumeft, d_volumeftasym, dimsvolume);
	hipFree(d_volumeft);

	//tcomplex* h_volumeft = (tcomplex*)MallocFromDeviceArray(d_volumeft, ElementsFFT(dimsvolume) * sizeof(tcomplex));
	//free(h_volumeft);
	//d_MultiplyByScalar((tfloat*)d_volumeft, (tfloat*)d_volumeft, ElementsFFT(dimsvolume) * 2, (tfloat)1 / (tfloat)dimsproj.z);
	d_IFFTC2R(d_volumeftasym, d_volume, 3, dimsvolume);

	d_RemapFullFFT2Full(d_volume, d_volume, dimsvolume);


	hipFree(d_volumeftasym);
}


////////////////
//CUDA kernels//
////////////////

__global__ void ReconstructFourierKernel(tcomplex* d_projft, tcomplex* d_volumeft, tfloat* d_samples, int3 dimsvolume, int3 dimsproj, glm::vec3* d_vecX, glm::vec3* d_vecY)
{
	int elements = Elements(dimsproj);
	d_projft += elements * blockIdx.y;

	for (int id = blockIdx.x * blockDim.x + threadIdx.x; id < elements; id += gridDim.x * blockDim.x)
	{
		int y = id / (dimsproj.x);
		int x = id % (dimsproj.x);

		glm::vec3 rotated = (float)(x - dimsvolume.x / 2) * d_vecX[blockIdx.y] + (float)(y - dimsvolume.y / 2) * d_vecY[blockIdx.y];
		if(rotated.x * rotated.x + rotated.y * rotated.y + rotated.z * rotated.z >= dimsvolume.x * dimsvolume.x / 4)
			continue;

		//bool isnegative = false;
		/*if(rotated.x > 0.0f)
		{
			rotated = -rotated;
			isnegative = true;
		}*/
		rotated += glm::vec3((float)(dimsvolume.x / 2));
		int x0 = (int)rotated.x;
		int y0 = (int)rotated.y;
		int z0 = (int)rotated.z;

		if(x0 >= dimsvolume.x || y0 >= dimsvolume.y || z0 >= dimsvolume.z)
			continue;

		int x1 = min(x0 + 1, dimsvolume.x - 1);
		int y1 = min(y0 + 1, dimsvolume.y - 1);
		int z1 = min(z0 + 1, dimsvolume.z - 1);

		tcomplex val = d_projft[id];
		//if(isnegative)
			//val = cconj(val);
		
		float xd = rotated.x - floor(rotated.x);
		float yd = rotated.y - floor(rotated.y);
		float zd = rotated.z - floor(rotated.z);

		float c0 = 1.0f - zd;
		float c1 = zd;

		float c00 = (1.0f - yd) * c0;
		float c10 = yd * c0;
		float c01 = (1.0f - yd) * c1;
		float c11 = yd * c1;

		float c000 = (1.0f - xd) * c00;
		float c100 = xd * c00;
		float c010 = (1.0f - xd) * c10;
		float c110 = xd * c10;
		float c001 = (1.0f - xd) * c01;
		float c101 = xd * c01;
		float c011 = (1.0f - xd) * c11;
		float c111 = xd * c11;

		atomicAdd((tfloat*)(d_volumeft + (z0 * dimsvolume.y + y0) * (dimsvolume.x) + x0), c000 * val.x);
		atomicAdd(((tfloat*)(d_volumeft + (z0 * dimsvolume.y + y0) * (dimsvolume.x) + x0)) + 1, c000 * val.y);
		atomicAdd((tfloat*)(d_samples + (z0 * dimsvolume.y + y0) * (dimsvolume.x) + x0), c000);

		atomicAdd((tfloat*)(d_volumeft + (z0 * dimsvolume.y + y0) * (dimsvolume.x) + x1), c100 * val.x);
		atomicAdd(((tfloat*)(d_volumeft + (z0 * dimsvolume.y + y0) * (dimsvolume.x) + x1)) + 1, c100 * val.y);
		atomicAdd((tfloat*)(d_samples + (z0 * dimsvolume.y + y0) * (dimsvolume.x) + x1), c100);

		atomicAdd((tfloat*)(d_volumeft + (z0 * dimsvolume.y + y1) * (dimsvolume.x) + x0), c010 * val.x);
		atomicAdd(((tfloat*)(d_volumeft + (z0 * dimsvolume.y + y1) * (dimsvolume.x) + x0)) + 1, c010 * val.y);
		atomicAdd((tfloat*)(d_samples + (z0 * dimsvolume.y + y1) * (dimsvolume.x) + x0), c010);

		atomicAdd((tfloat*)(d_volumeft + (z0 * dimsvolume.y + y1) * (dimsvolume.x) + x1), c110 * val.x);
		atomicAdd(((tfloat*)(d_volumeft + (z0 * dimsvolume.y + y1) * (dimsvolume.x) + x1)) + 1, c110 * val.y);
		atomicAdd((tfloat*)(d_samples + (z0 * dimsvolume.y + y1) * (dimsvolume.x) + x1), c110);


		atomicAdd((tfloat*)(d_volumeft + (z1 * dimsvolume.y + y0) * (dimsvolume.x) + x0), c001 * val.x);
		atomicAdd(((tfloat*)(d_volumeft + (z1 * dimsvolume.y + y0) * (dimsvolume.x) + x0)) + 1, c001 * val.y);
		atomicAdd((tfloat*)(d_samples + (z1 * dimsvolume.y + y0) * (dimsvolume.x) + x0), c001);

		atomicAdd((tfloat*)(d_volumeft + (z1 * dimsvolume.y + y0) * (dimsvolume.x) + x1), c101 * val.x);
		atomicAdd(((tfloat*)(d_volumeft + (z1 * dimsvolume.y + y0) * (dimsvolume.x) + x1)) + 1, c101 * val.y);
		atomicAdd((tfloat*)(d_samples + (z1 * dimsvolume.y + y0) * (dimsvolume.x) + x1), c101);

		atomicAdd((tfloat*)(d_volumeft + (z1 * dimsvolume.y + y1) * (dimsvolume.x) + x0), c011 * val.x);
		atomicAdd(((tfloat*)(d_volumeft + (z1 * dimsvolume.y + y1) * (dimsvolume.x) + x0)) + 1, c011 * val.y);
		atomicAdd((tfloat*)(d_samples + (z1 * dimsvolume.y + y1) * (dimsvolume.x) + x0), c011);

		atomicAdd((tfloat*)(d_volumeft + (z1 * dimsvolume.y + y1) * (dimsvolume.x) + x1), c111 * val.x);
		atomicAdd(((tfloat*)(d_volumeft + (z1 * dimsvolume.y + y1) * (dimsvolume.x) + x1)) + 1, c111 * val.y);
		atomicAdd((tfloat*)(d_samples + (z1 * dimsvolume.y + y1) * (dimsvolume.x) + x1), c111);
	}
}