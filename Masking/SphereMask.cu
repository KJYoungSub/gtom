#include "hip/hip_runtime.h"
#include "..\Prerequisites.cuh"
#include "..\Functions.cuh"


////////////////////////////
//CUDA kernel declarations//
////////////////////////////

template <class T> __global__ void SphereMaskKernel(T const* const d_input, T* const d_output, int3 const size, tfloat const radius, tfloat const sigma, tfloat3 center);


////////////////
//Host methods//
////////////////

template <class T> void d_SphereMask(T const* const d_input, 
									 T* const d_output, 
									 int3 const size, 
									 tfloat const* const radius,
									 tfloat const sigma,
									 tfloat3 const* const center,
									 int batch)
{
	tfloat _radius = radius != NULL ? *radius : (min(min(size.x, size.y), size.z) - 1) / 2;
	tfloat3 _center = center != NULL ? *center : tfloat3(size.x / 2 + 1, size.y / 2 + 1, size.z / 2 + 1);

	int TpB = 256;
	dim3 grid = dim3(size.y, size.z, batch);
	SphereMaskKernel<T> <<<grid, TpB>>> (d_input, d_output, size, _radius, sigma, _center);

	hipDeviceSynchronize();
}
template void d_SphereMask<tfloat>(tfloat const* const d_input, tfloat* const d_output, int3 const size, tfloat const* const radius, tfloat const sigma, tfloat3 const* const center, int batch);
//template void d_SphereMask<tcomplex>(tcomplex const* const d_input, tcomplex* const d_output, int3 const size, tfloat const radius, tfloat const sigma, tfloat3 const center, int batch);


////////////////
//CUDA kernels//
////////////////

template <class T> __global__ void SphereMaskKernel(T const* const d_input, T* const d_output, int3 const size, tfloat const radius, tfloat const sigma, tfloat3 center)
{
	if(threadIdx.x >= size.x)
		return;

	//For batch mode
	int offset = blockIdx.z * size.x * size.y * size.z + blockIdx.y * size.x * size.y + blockIdx.x * size.x;

	tfloat xsq, ysq, zsq, length;
	T maskvalue;
	
	//Squared y and z distance from center
	ysq = (tfloat)(blockIdx.x + 1) - center.y;
	ysq *= ysq;
	if(size.z > 1)
	{
		zsq = (tfloat)(blockIdx.y + 1) - center.z;
		zsq *= zsq;
	}
	else
		zsq = 0;

	for(int x = threadIdx.x; x < size.x; x += blockDim.x)
	{
		xsq = (tfloat)(x + 1) - center.x;
		xsq *= xsq;
		//Distance from center
		length = sqrt(xsq + ysq + zsq);

		if(length < radius)
			maskvalue = 1;
		else
		{
			//Smooth border
			if(sigma > (tfloat)0)
			{
				maskvalue = exp(-((length - radius) * (length - radius) / (sigma * sigma)));
				if(maskvalue < (tfloat)0.1353)
					maskvalue = 0;
			}
			//Hard border
			else
				maskvalue = 0;
		}

		//Write masked input to output
		d_output[offset + x] = maskvalue * d_input[offset + x];
	}
}