#include "hip/hip_runtime.h"
#include "../Prerequisites.cuh"
#include "../Functions.cuh"


////////////////////////////
//CUDA kernel declarations//
////////////////////////////

template <class T> __global__ void MultiplyByVectorKernel(T* d_input, T* multiplicators, T* d_output, size_t elements, int batch);
template <class T> __global__ void MultiplyByScalarKernel(T* d_input, T* d_output, size_t elements, T multiplicator);
template <class T> __global__ void MultiplyByScalarKernel(T* d_input, T* multiplicators, T* d_output, size_t elements);

__global__ void ComplexMultiplyByVectorKernel(tcomplex* d_input, tfloat* multiplicators, tcomplex* d_output, size_t elements, int batch);
__global__ void ComplexMultiplyByScalarKernel(tcomplex* d_input, tcomplex* d_output, size_t elements, tfloat multiplicator);
__global__ void ComplexMultiplyByScalarKernel(tcomplex* d_input, tfloat* multiplicators, tcomplex* d_output, size_t elements);

__global__ void ComplexMultiplyByVectorKernel(tcomplex* d_input, tcomplex* multiplicators, tcomplex* d_output, size_t elements, int batch);
__global__ void ComplexMultiplyByScalarKernel(tcomplex* d_input, tcomplex* d_output, size_t elements, tcomplex multiplicator);
__global__ void ComplexMultiplyByScalarKernel(tcomplex* d_input, tcomplex* multiplicators, tcomplex* d_output, size_t elements);

__global__ void ComplexMultiplyByConjVectorKernel(tcomplex* d_input, tcomplex* multiplicators, tcomplex* d_output, size_t elements, int batch);
__global__ void ComplexMultiplyByConjScalarKernel(tcomplex* d_input, tcomplex* d_output, size_t elements, tcomplex multiplicator);
__global__ void ComplexMultiplyByConjScalarKernel(tcomplex* d_input, tcomplex* multiplicators, tcomplex* d_output, size_t elements);

template <class T> __global__ void AddVectorKernel(T* d_input, T* d_summands, T* d_output, size_t elements, int batch);
template <class T> __global__ void AddScalarKernel(T* d_input, T* d_output, size_t elements, T summand);
template <class T> __global__ void AddScalarKernel(T* d_input, T* d_summands, T* d_output, size_t elements);

template <class T> __global__ void SubtractVectorKernel(T* d_input, T* d_subtrahends, T* d_output, size_t elements, int batch);
template <class T> __global__ void SubtractScalarKernel(T* d_input, T* d_output, size_t elements, T subtrahend);
template <class T> __global__ void SubtractScalarKernel(T* d_input, T* d_subtrahends, T* d_output, size_t elements);

template <class T> __global__ void SquareKernel(T* d_input, T* d_output, size_t elements);
template <class T> __global__ void SqrtKernel(T* d_input, T* d_output, size_t elements);
template <class T> __global__ void PowKernel(T* d_input, T* d_output, size_t elements, T exponent);
template <class T> __global__ void AbsKernel(T* d_input, T* d_output, size_t elements);

template <class T> __global__ void MaxOpKernel(T* d_input1, T* d_input2, T* d_output, size_t elements);
template <class T> __global__ void MinOpKernel(T* d_input1, T* d_input2, T* d_output, size_t elements);


//////////////////
//Multiplication//
//////////////////

template <class T> void d_MultiplyByVector(T* d_input, T* d_multiplicators, T* d_output, size_t elements, int batch)
{
	size_t TpB = min(256, elements);
	size_t totalblocks = min((elements + TpB - 1) / TpB, 32768);
	dim3 grid = dim3((uint)totalblocks);
	MultiplyByVectorKernel<T> <<<grid, (uint)TpB>>> (d_input, d_multiplicators, d_output, elements, batch);
	hipStreamQuery(0);
}
template void d_MultiplyByVector<tfloat>(tfloat* d_input, tfloat* d_multiplicators, tfloat* d_output, size_t elements, int batch);
template void d_MultiplyByVector<int>(int* d_input, int* d_multiplicators, int* d_output, size_t elements, int batch);

template <class T> void d_MultiplyByScalar(T* d_input, T* d_output, size_t elements, T multiplicator)
{
	size_t TpB = min(256, elements);
	size_t totalblocks = min((elements + TpB - 1) / TpB, 32768);
	dim3 grid = dim3((uint)totalblocks);
	MultiplyByScalarKernel<T> <<<grid, (uint)TpB>>> (d_input, d_output, elements, multiplicator);
	hipStreamQuery(0);
}
template void d_MultiplyByScalar<tfloat>(tfloat* d_input, tfloat* d_output, size_t elements, tfloat multiplicator);
template void d_MultiplyByScalar<double>(double* d_input, double* d_output, size_t elements, double multiplicator);
template void d_MultiplyByScalar<int>(int* d_input, int* d_output, size_t elements, int multiplicator);

template <class T> void d_MultiplyByScalar(T* d_input, T* d_multiplicators, T* d_output, size_t elements, int batch)
{
	size_t TpB = min(256, elements);
	size_t totalblocks = min((elements + TpB - 1) / TpB, 32768);
	dim3 grid = dim3((uint)totalblocks, batch);
	MultiplyByScalarKernel<T> <<<grid, (uint)TpB>>> (d_input, d_multiplicators, d_output, elements);
	hipStreamQuery(0);
}
template void d_MultiplyByScalar<tfloat>(tfloat* d_input, tfloat* d_multiplicators, tfloat* d_output, size_t elements, int batch);
template void d_MultiplyByScalar<int>(int* d_input, int* d_multiplicators, int* d_output, size_t elements, int batch);

template <class T> __global__ void MultiplyByVectorKernel(T* d_input, T* d_multiplicators, T* d_output, size_t elements, int batch)
{
	T val;
	for(size_t id = blockIdx.x * blockDim.x + threadIdx.x; 
		id < elements; 
		id += blockDim.x * gridDim.x)
	{
		val = d_multiplicators[id];
		for(size_t n = 0; n < batch; n++)
			d_output[id + elements * n] = d_input[id + elements * n] * val;
	}
}

template <class T> __global__ void MultiplyByScalarKernel(T* d_input, T* d_output, size_t elements, T multiplicator)
{
	for(size_t id = blockIdx.x * blockDim.x + threadIdx.x; 
		id < elements; 
		id += blockDim.x * gridDim.x)
		d_output[id] = d_input[id] * multiplicator;
}

template <class T> __global__ void MultiplyByScalarKernel(T* d_input, T* d_multiplicators, T* d_output, size_t elements)
{
	__shared__ T scalar;
	if(threadIdx.x == 0)
		scalar = d_multiplicators[blockIdx.y];
	__syncthreads();

	size_t offset = elements * blockIdx.y;
	for(size_t id = blockIdx.x * blockDim.x + threadIdx.x; 
		id < elements; 
		id += blockDim.x * gridDim.x)
		d_output[id + offset] = d_input[id + offset] * scalar;
}


//////////////////////////
//Complex Multiplication//
//////////////////////////

void d_ComplexMultiplyByVector(tcomplex* d_input, tfloat* d_multiplicators, tcomplex* d_output, size_t elements, int batch)
{
	size_t TpB = min(256, elements);
	size_t totalblocks = min((elements + TpB - 1) / TpB, 32768);
	dim3 grid = dim3((uint)totalblocks);
	ComplexMultiplyByVectorKernel <<<grid, (uint)TpB>>> (d_input, d_multiplicators, d_output, elements, batch);
	hipStreamQuery(0);
}

void d_ComplexMultiplyByVector(tcomplex* d_input, tcomplex* d_multiplicators, tcomplex* d_output, size_t elements, int batch)
{
	size_t TpB = min(256, elements);
	size_t totalblocks = min((elements + TpB - 1) / TpB, 32768);
	dim3 grid = dim3((uint)totalblocks);
	ComplexMultiplyByVectorKernel <<<grid, (uint)TpB>>> (d_input, d_multiplicators, d_output, elements, batch);
	hipStreamQuery(0);
}

void d_ComplexMultiplyByConjVector(tcomplex* d_input, tcomplex* d_multiplicators, tcomplex* d_output, size_t elements, int batch)
{
	size_t TpB = min(256, elements);
	size_t totalblocks = min((elements + TpB - 1) / TpB, 32768);
	dim3 grid = dim3((uint)totalblocks);
	ComplexMultiplyByConjVectorKernel <<<grid, (uint)TpB>>> (d_input, d_multiplicators, d_output, elements, batch);
	hipStreamQuery(0);
}

void d_ComplexMultiplyByScalar(tcomplex* d_input, tcomplex* d_output, size_t elements, tfloat multiplicator)
{
	size_t TpB = min(256, elements);
	size_t totalblocks = min((elements + TpB - 1) / TpB, 32768);
	dim3 grid = dim3((uint)totalblocks);
	ComplexMultiplyByScalarKernel <<<grid, (uint)TpB>>> (d_input, d_output, elements, multiplicator);
	hipStreamQuery(0);
}

void d_ComplexMultiplyByScalar(tcomplex* d_input, tcomplex* d_output, size_t elements, tcomplex multiplicator)
{
	size_t TpB = min(256, elements);
	size_t totalblocks = min((elements + TpB - 1) / TpB, 32768);
	dim3 grid = dim3((uint)totalblocks);
	ComplexMultiplyByScalarKernel <<<grid, (uint)TpB>>> (d_input, d_output, elements, multiplicator);
	hipStreamQuery(0);
}

void d_ComplexMultiplyByConjScalar(tcomplex* d_input, tcomplex* d_output, size_t elements, tcomplex multiplicator)
{
	size_t TpB = min(256, elements);
	size_t totalblocks = min((elements + TpB - 1) / TpB, 32768);
	dim3 grid = dim3((uint)totalblocks);
	ComplexMultiplyByConjScalarKernel <<<grid, (uint)TpB>>> (d_input, d_output, elements, multiplicator);
	hipStreamQuery(0);
}

void d_ComplexMultiplyByScalar(tcomplex* d_input, tfloat* d_multiplicators, tcomplex* d_output, size_t elements, int batch)
{
	size_t TpB = min(256, elements);
	size_t totalblocks = min((elements + TpB - 1) / TpB, 32768);
	dim3 grid = dim3((uint)totalblocks, batch);
	ComplexMultiplyByScalarKernel <<<grid, (uint)TpB>>> (d_input, d_multiplicators, d_output, elements);
	hipStreamQuery(0);
}

void d_ComplexMultiplyByScalar(tcomplex* d_input, tcomplex* d_multiplicators, tcomplex* d_output, size_t elements, int batch)
{
	size_t TpB = min(256, elements);
	size_t totalblocks = min((elements + TpB - 1) / TpB, 32768);
	dim3 grid = dim3((uint)totalblocks, batch);
	ComplexMultiplyByScalarKernel <<<grid, (uint)TpB>>> (d_input, d_multiplicators, d_output, elements);
	hipStreamQuery(0);
}

void d_ComplexMultiplyByConjScalar(tcomplex* d_input, tcomplex* d_multiplicators, tcomplex* d_output, size_t elements, int batch)
{
	size_t TpB = min(256, elements);
	size_t totalblocks = min((elements + TpB - 1) / TpB, 32768);
	dim3 grid = dim3((uint)totalblocks, batch);
	ComplexMultiplyByConjScalarKernel <<<grid, (uint)TpB>>> (d_input, d_multiplicators, d_output, elements);
	hipStreamQuery(0);
}

__global__ void ComplexMultiplyByVectorKernel(tcomplex* d_input, tfloat* d_multiplicators, tcomplex* d_output, size_t elements, int batch)
{
	tfloat val;
	for(size_t id = blockIdx.x * blockDim.x + threadIdx.x; 
		id < elements; 
		id += blockDim.x * gridDim.x)
	{
		val = d_multiplicators[id];
		for(size_t n = 0; n < batch; n++)
		{
			d_output[id + elements * n].x = d_input[id + elements * n].x * val;
			d_output[id + elements * n].y = d_input[id + elements * n].y * val;
		}
	}
}

__global__ void ComplexMultiplyByVectorKernel(tcomplex* d_input, tcomplex* d_multiplicators, tcomplex* d_output, size_t elements, int batch)
{
	tcomplex val;
	for(size_t id = blockIdx.x * blockDim.x + threadIdx.x; 
		id < elements; 
		id += blockDim.x * gridDim.x)
	{
		val = d_multiplicators[id];
		for(size_t n = 0; n < batch; n++)
		{
			d_output[id + elements * n] = cmul(d_input[id + elements * n], val);
		}
	}
}

__global__ void ComplexMultiplyByConjVectorKernel(tcomplex* d_input, tcomplex* d_multiplicators, tcomplex* d_output, size_t elements, int batch)
{
	tcomplex val;
	for(size_t id = blockIdx.x * blockDim.x + threadIdx.x; 
		id < elements; 
		id += blockDim.x * gridDim.x)
	{
		val = d_multiplicators[id];
		for(size_t n = 0; n < batch; n++)
		{
			d_output[id + elements * n] = cmul(d_input[id + elements * n], cconj(val));
		}
	}
}

__global__ void ComplexMultiplyByScalarKernel(tcomplex* d_input, tcomplex* d_output, size_t elements, tfloat multiplicator)
{
	for(size_t id = blockIdx.x * blockDim.x + threadIdx.x; 
		id < elements; 
		id += blockDim.x * gridDim.x)
	{
		d_output[id].x = d_input[id].x * multiplicator;
		d_output[id].y = d_input[id].y * multiplicator;
	}
}

__global__ void ComplexMultiplyByScalarKernel(tcomplex* d_input, tcomplex* d_output, size_t elements, tcomplex multiplicator)
{
	for(size_t id = blockIdx.x * blockDim.x + threadIdx.x; 
		id < elements; 
		id += blockDim.x * gridDim.x)
		d_output[id] = cmul(d_input[id], multiplicator);
}

__global__ void ComplexMultiplyByConjScalarKernel(tcomplex* d_input, tcomplex* d_output, size_t elements, tcomplex multiplicator)
{
	for(size_t id = blockIdx.x * blockDim.x + threadIdx.x; 
		id < elements; 
		id += blockDim.x * gridDim.x)
		d_output[id] = cmul(d_input[id], cconj(multiplicator));
}

__global__ void ComplexMultiplyByScalarKernel(tcomplex* d_input, tfloat* d_multiplicators, tcomplex* d_output, size_t elements)
{
	__shared__ tfloat scalar;
	if(threadIdx.x == 0)
		scalar = d_multiplicators[blockIdx.y];
	__syncthreads();

	size_t offset = elements * blockIdx.y;
	for(size_t id = blockIdx.x * blockDim.x + threadIdx.x; 
		id < elements; 
		id += blockDim.x * gridDim.x)
	{
		d_output[id + offset].x = d_input[id + offset].x * scalar;
		d_output[id + offset].y = d_input[id + offset].y * scalar;
	}
}

__global__ void ComplexMultiplyByScalarKernel(tcomplex* d_input, tcomplex* d_multiplicators, tcomplex* d_output, size_t elements)
{
	__shared__ tcomplex scalar;
	if(threadIdx.x == 0)
		scalar = d_multiplicators[blockIdx.y];
	__syncthreads();

	size_t offset = elements * blockIdx.y;
	for(size_t id = blockIdx.x * blockDim.x + threadIdx.x; 
		id < elements; 
		id += blockDim.x * gridDim.x)
		d_output[id + offset] = cmul(d_input[id + offset], scalar);
}

__global__ void ComplexMultiplyByConjScalarKernel(tcomplex* d_input, tcomplex* d_multiplicators, tcomplex* d_output, size_t elements)
{
	__shared__ tcomplex scalar;
	if(threadIdx.x == 0)
		scalar = d_multiplicators[blockIdx.y];
	__syncthreads();

	size_t offset = elements * blockIdx.y;
	for(size_t id = blockIdx.x * blockDim.x + threadIdx.x; 
		id < elements; 
		id += blockDim.x * gridDim.x)
		d_output[id + offset] = cmul(d_input[id + offset], cconj(scalar));
}


////////////
//Addition//
////////////

template <class T> void d_AddVector(T* d_input, T* d_summands, T* d_output, size_t elements, int batch)
{
	size_t TpB = min(256, elements);
	size_t totalblocks = min((elements + TpB - 1) / TpB, 32768);
	dim3 grid = dim3((uint)totalblocks);
	AddVectorKernel<T> <<<grid, (uint)TpB>>> (d_input, d_summands, d_output, elements, batch);
}
template void d_AddVector<tfloat>(tfloat* d_input, tfloat* d_summands, tfloat* d_output, size_t elements, int batch);
template void d_AddVector<int>(int* d_input, int* d_summands, int* d_output, size_t elements, int batch);

template <class T> void d_AddScalar(T* d_input, T* d_output, size_t elements, T summand)
{
	size_t TpB = min(256, elements);
	size_t totalblocks = min((elements + TpB - 1) / TpB, 32768);
	dim3 grid = dim3((uint)totalblocks);
	AddScalarKernel<T> <<<grid, (uint)TpB>>> (d_input, d_output, elements, summand);
}
template void d_AddScalar<tfloat>(tfloat* d_input, tfloat* d_output, size_t elements, tfloat summand);
template void d_AddScalar<int>(int* d_input, int* d_output, size_t elements, int summand);

template <class T> void d_AddScalar(T* d_input, T* d_summands, T* d_output, size_t elements, int batch)
{
	size_t TpB = min(256, elements);
	size_t totalblocks = min((elements + TpB - 1) / TpB, 32768);
	dim3 grid = dim3((uint)totalblocks, batch);
	AddScalarKernel<T> <<<grid, (uint)TpB>>> (d_input, d_summands, d_output, elements);
}
template void d_AddScalar<tfloat>(tfloat* d_input, tfloat* d_summands, tfloat* d_output, size_t elements, int batch);
template void d_AddScalar<int>(int* d_input, int* d_summands, int* d_output, size_t elements, int batch);

template <class T> __global__ void AddVectorKernel(T* d_input, T* d_summands, T* d_output, size_t elements, int batch)
{
	T val;
	for(size_t id = blockIdx.x * blockDim.x + threadIdx.x; 
		id < elements; 
		id += blockDim.x * gridDim.x)
	{
		val = d_summands[id];
		for(size_t n = 0; n < batch; n++)
			d_output[id + elements * n] = d_input[id + elements * n] + val;
	}
}

template <class T> __global__ void AddScalarKernel(T* d_input, T* d_output, size_t elements, T summand)
{
	for(size_t id = blockIdx.x * blockDim.x + threadIdx.x; 
		id < elements; 
		id += blockDim.x * gridDim.x)
		d_output[id] = d_input[id] + summand;
}

template <class T> __global__ void AddScalarKernel(T* d_input, T* d_summands, T* d_output, size_t elements)
{
	__shared__ T scalar;
	if(threadIdx.x == 0)
		scalar = d_summands[blockIdx.y];
	__syncthreads();

	size_t offset = elements * blockIdx.y;
	for(size_t id = blockIdx.x * blockDim.x + threadIdx.x; 
		id < elements; 
		id += blockDim.x * gridDim.x)
		d_output[id + offset] = d_input[id + offset] + scalar;
}


///////////////
//Subtraction//
///////////////

template <class T> void d_SubtractVector(T* d_input, T* d_subtrahends, T* d_output, size_t elements, int batch)
{
	size_t TpB = min(256, elements);
	size_t totalblocks = min((elements + TpB - 1) / TpB, 32768);
	dim3 grid = dim3((uint)totalblocks);
	SubtractVectorKernel<T> <<<grid, (uint)TpB>>> (d_input, d_subtrahends, d_output, elements, batch);
}
template void d_SubtractVector<tfloat>(tfloat* d_input, tfloat* d_subtrahends, tfloat* d_output, size_t elements, int batch);
template void d_SubtractVector<int>(int* d_input, int* d_subtrahends, int* d_output, size_t elements, int batch);

template <class T> void d_SubtractScalar(T* d_input, T* d_output, size_t elements, T subtrahend)
{
	size_t TpB = min(256, elements);
	size_t totalblocks = min((elements + TpB - 1) / TpB, 32768);
	dim3 grid = dim3((uint)totalblocks);
	SubtractScalarKernel<T> <<<grid, (uint)TpB>>> (d_input, d_output, elements, subtrahend);
}
template void d_SubtractScalar<tfloat>(tfloat* d_input, tfloat* d_output, size_t elements, tfloat subtrahend);
template void d_SubtractScalar<int>(int* d_input, int* d_output, size_t elements, int subtrahend);

template <class T> void d_SubtractScalar(T* d_input, T* d_subtrahends, T* d_output, size_t elements, int batch)
{
	size_t TpB = min(256, elements);
	size_t totalblocks = min((elements + TpB - 1) / TpB, 32768);
	dim3 grid = dim3((uint)totalblocks, batch);
	SubtractScalarKernel<T> <<<grid, (uint)TpB>>> (d_input, d_subtrahends, d_output, elements);
}
template void d_SubtractScalar<tfloat>(tfloat* d_input, tfloat* d_subtrahends, tfloat* d_output, size_t elements, int batch);
template void d_SubtractScalar<int>(int* d_input, int* d_subtrahends, int* d_output, size_t elements, int batch);

template <class T> __global__ void SubtractVectorKernel(T* d_input, T* d_subtrahends, T* d_output, size_t elements, int batch)
{
	T val;
	for(size_t id = blockIdx.x * blockDim.x + threadIdx.x; 
		id < elements; 
		id += blockDim.x * gridDim.x)
	{
		val = d_subtrahends[id];
		for(size_t n = 0; n < batch; n++)
			d_output[id + elements * n] = d_input[id + elements * n] - val;
	}
}

template <class T> __global__ void SubtractScalarKernel(T* d_input, T* d_output, size_t elements, T subtrahend)
{
	for(size_t id = blockIdx.x * blockDim.x + threadIdx.x; 
		id < elements; 
		id += blockDim.x * gridDim.x)
		d_output[id] = d_input[id] - subtrahend;
}

template <class T> __global__ void SubtractScalarKernel(T* d_input, T* d_subtrahends, T* d_output, size_t elements)
{
	__shared__ T scalar;
	if(threadIdx.x == 0)
		scalar = d_subtrahends[blockIdx.y];
	__syncthreads();

	size_t offset = elements * blockIdx.y;
	size_t gridsize = blockDim.x * gridDim.x;
	for(size_t id = blockIdx.x * blockDim.x + threadIdx.x; 
		id < elements; 
		id += gridsize)
		d_output[id + offset] = d_input[id + offset] - scalar;
}


//////////
//Square//
//////////

template <class T> void d_Square(T* d_input, T* d_output, size_t elements, int batch)
{
	size_t TpB = min(256, elements);
	size_t totalblocks = min((elements + TpB - 1) / TpB, 32768);
	dim3 grid = dim3((uint)totalblocks);
	for(int b = 0; b < batch; b++)
		SquareKernel<T> <<<grid, (uint)TpB>>> (d_input + elements * b, d_output + elements * b, elements);
}
template void d_Square<tfloat>(tfloat* d_input, tfloat* d_output, size_t elements, int batch);
template void d_Square<int>(int* d_input, int* d_output, size_t elements, int batch);

template <class T> __global__ void SquareKernel(T* d_input, T* d_output, size_t elements)
{
	T val;
	int gridsize = blockDim.x * gridDim.x;
	for(int id = blockIdx.x * blockDim.x + threadIdx.x; 
		id < elements; 
		id += gridsize)
	{
		val = d_input[id];
		d_output[id] = val * val;
	}
}


///////////////
//Square root//
///////////////

template <class T> void d_Sqrt(T* d_input, T* d_output, size_t elements)
{
	size_t TpB = min(256, elements);
	size_t totalblocks = min((elements + TpB - 1) / TpB, 8192);
	dim3 grid = dim3((uint)totalblocks);
	SqrtKernel<T> <<<grid, (uint)TpB>>> (d_input, d_output, elements);
}
template void d_Sqrt<tfloat>(tfloat* d_input, tfloat* d_output, size_t elements);
//template void d_Sqrt<int>(int* d_input, int* d_output, size_t elements);

template <class T> __global__ void SqrtKernel(T* d_input, T* d_output, size_t elements)
{
	for(size_t id = blockIdx.x * blockDim.x + threadIdx.x; 
		id < elements; 
		id += blockDim.x * gridDim.x)
		d_output[id] = sqrt(d_input[id]);
}


/////////
//Power//
/////////

template <class T> void d_Pow(T* d_input, T* d_output, size_t elements, T exponent)
{
	size_t TpB = min(256, elements);
	size_t totalblocks = min((elements + TpB - 1) / TpB, 8192);
	dim3 grid = dim3((uint)totalblocks);
	PowKernel<T> <<<grid, (uint)TpB>>> (d_input, d_output, elements, exponent);
}
template void d_Pow<tfloat>(tfloat* d_input, tfloat* d_output, size_t elements, tfloat exponent);

template <class T> __global__ void PowKernel(T* d_input, T* d_output, size_t elements, T exponent)
{
	for(size_t id = blockIdx.x * blockDim.x + threadIdx.x; 
		id < elements; 
		id += blockDim.x * gridDim.x)
		d_output[id] = pow(d_input[id], exponent);
}


///////
//Abs//
///////

template <class T> void d_Abs(T* d_input, T* d_output, size_t elements)
{
	size_t TpB = min(256, elements);
	size_t totalblocks = min((elements + TpB - 1) / TpB, 8192);
	dim3 grid = dim3((uint)totalblocks);
	AbsKernel<T> <<<grid, (uint)TpB>>> (d_input, d_output, elements);
}
template void d_Abs<tfloat>(tfloat* d_input, tfloat* d_output, size_t elements);

template <class T> __global__ void AbsKernel(T* d_input, T* d_output, size_t elements)
{
	for(size_t id = blockIdx.x * blockDim.x + threadIdx.x; 
		id < elements; 
		id += blockDim.x * gridDim.x)
		d_output[id] = abs(d_input[id]);
}


///////////////
//Min/Max ops//
///////////////

template <class T> void d_MaxOp(T* d_input1, T* d_input2, T* d_output, size_t elements)
{
	size_t TpB = min(256, elements);
	size_t totalblocks = min((elements + TpB - 1) / TpB, 8192);
	dim3 grid = dim3((uint)totalblocks);
	MaxOpKernel<T> <<<grid, (uint)TpB>>> (d_input1, d_input2, d_output, elements);
}
template void d_MaxOp<int>(int* d_input1, int* d_input2, int* d_output, size_t elements);
template void d_MaxOp<float>(float* d_input1, float* d_input2, float* d_output, size_t elements);
template void d_MaxOp<double>(double* d_input1, double* d_input2, double* d_output, size_t elements);

template <class T> void d_MinOp(T* d_input1, T* d_input2, T* d_output, size_t elements)
{
	size_t TpB = min(256, elements);
	size_t totalblocks = min((elements + TpB - 1) / TpB, 8192);
	dim3 grid = dim3((uint)totalblocks);
	MinOpKernel<T> <<<grid, (uint)TpB>>> (d_input1, d_input2, d_output, elements);
}
template void d_MinOp<int>(int* d_input1, int* d_input2, int* d_output, size_t elements);
template void d_MinOp<float>(float* d_input1, float* d_input2, float* d_output, size_t elements);
template void d_MinOp<double>(double* d_input1, double* d_input2, double* d_output, size_t elements);

template <class T> __global__ void MaxOpKernel(T* d_input1, T* d_input2, T* d_output, size_t elements)
{
	for(size_t id = blockIdx.x * blockDim.x + threadIdx.x; 
		id < elements; 
		id += blockDim.x * gridDim.x)
		d_output[id] = max(d_input1[id], d_input2[id]);
}

template <class T> __global__ void MinOpKernel(T* d_input1, T* d_input2, T* d_output, size_t elements)
{
	for(size_t id = blockIdx.x * blockDim.x + threadIdx.x; 
		id < elements; 
		id += blockDim.x * gridDim.x)
		d_output[id] = min(d_input1[id], d_input2[id]);
}


////////
//Misc//
////////

size_t NextPow2(size_t x)
{
    --x;
    x |= x >> 1;
    x |= x >> 2;
    x |= x >> 4;
    x |= x >> 8;
    x |= x >> 16;
    return ++x;
}

bool IsPow2(size_t x) 
{
	return x && !(x & (x - 1));
}