#include "hip/hip_runtime.h"
#include "Prerequisites.cuh"
#include "Angles.cuh"
#include "DeviceFunctions.cuh"
#include "Helper.cuh"


////////////////////////////
//CUDA kernel declarations//
////////////////////////////

template <bool iscentered> __global__ void Exact2DWeightingKernel(tfloat* d_weights, int2 dims, glm::vec3* d_normals, glm::mat3x2* d_globalB2localB, tfloat maxfreq);
template <bool iscentered> __global__ void Exact3DWeightingKernel(tfloat* d_weights, int3 dims, glm::vec3* d_normals, int nimages, tfloat maxfreq);


/////////////////////////////////////////////////////////////////////////////////////
//2D weighting of frequency components for WBP reconstruction, using sinc(distance)//
/////////////////////////////////////////////////////////////////////////////////////

void d_Exact2DWeighting(tfloat* d_weights, int2 dimsimage, tfloat3* h_angles, int nimages, tfloat maxfreq, bool iszerocentered)
{
	glm::vec3* h_normals = (glm::vec3*)malloc(nimages * sizeof(glm::vec3));
	glm::mat3x2* h_globalB2localB = (glm::mat3x2*)malloc(nimages * sizeof(glm::mat3x2));

	for (int i = 0; i < nimages; i++)
	{
		glm::mat3 tB = Matrix3Euler(tfloat3(h_angles[i].x, h_angles[i].y, 0.0f));
		h_normals[i] = glm::vec3(tB[2][0], tB[2][1], tB[2][2]);
		h_globalB2localB[i] = glm::mat3x2(tB[0][0], tB[1][0], tB[0][1], tB[1][1], tB[0][2], tB[1][2]);	//Column-major layout in constructor
	}

	glm::vec3* d_normals = (glm::vec3*)CudaMallocFromHostArray(h_normals, nimages * sizeof(glm::vec3));
	glm::mat3x2* d_globalB2localB = (glm::mat3x2*)CudaMallocFromHostArray(h_globalB2localB, nimages * sizeof(glm::mat3x2));

	uint TpB = min(NextMultipleOf(dimsimage.x / 2 + 1, 32), 128);
	dim3 grid = dim3(dimsimage.y, nimages);
	if (iszerocentered)
		Exact2DWeightingKernel<true> <<<grid, TpB>>> (d_weights, dimsimage, d_normals, d_globalB2localB, maxfreq);
	else
		Exact2DWeightingKernel<false> <<<grid, TpB>>> (d_weights, dimsimage, d_normals, d_globalB2localB, maxfreq);

	free(h_globalB2localB);
	free(h_normals);
	hipFree(d_globalB2localB);
	hipFree(d_normals);
}

void d_Exact3DWeighting(tfloat* d_weights, int3 dimsvolume, tfloat3* h_angles, int nimages, tfloat maxfreq, bool iszerocentered)
{
	glm::vec3* h_normals = (glm::vec3*)malloc(nimages * sizeof(glm::vec3));

	for (int i = 0; i < nimages; i++)
	{
		glm::mat3 tB = Matrix3Euler(tfloat3(h_angles[i].x, h_angles[i].y, 0.0f));
		h_normals[i] = glm::vec3(tB[2][0], tB[2][1], tB[2][2]);
	}

	glm::vec3* d_normals = (glm::vec3*)CudaMallocFromHostArray(h_normals, nimages * sizeof(glm::vec3));

	uint TpB = min(NextMultipleOf(dimsvolume.x / 2 + 1, 32), 128);
	dim3 grid = dim3(dimsvolume.y, nimages);
	if (iszerocentered)
		Exact3DWeightingKernel<true> << <grid, TpB >> > (d_weights, dimsvolume, d_normals, nimages, maxfreq);
	else
		Exact3DWeightingKernel<false> << <grid, TpB >> > (d_weights, dimsvolume, d_normals, nimages, maxfreq);

	free(h_normals);
	hipFree(d_normals);
}


////////////////
//CUDA kernels//
////////////////

template <bool iscentered> __global__ void Exact2DWeightingKernel(tfloat* d_weights, int2 dims, glm::vec3* d_normals, glm::mat3x2* d_globalB2localB, tfloat maxfreq)
{
	int idy = blockIdx.x;
	int interpindex = blockIdx.y;

	int x, y;
	if (!iscentered)
		y = dims.y - 1 - ((idy + dims.y / 2 - 1) % dims.y);
	else
		y = idy;
	d_weights += y * (dims.x / 2 + 1);

	int elements = (dims.x / 2 + 1) * dims.y;
	glm::vec2 center = glm::vec2((float)(dims.x / 2), (float)(dims.y / 2));
	glm::vec3 normalA = d_normals[interpindex];

	for (int idx = threadIdx.x; idx < dims.x / 2 + 1; idx += blockDim.x)
	{
		if (!iscentered)
			x = dims.x / 2 - idx;
		else
			x = idx;

		glm::vec2 localA = glm::vec2((float)idx, (float)idy) - center;
		if (glm::length(localA) <= maxfreq)
		{
			glm::vec3 globalA = localA * d_globalB2localB[interpindex];
			float weightsum = 0.0f;

			for (int b = 0; b < gridDim.y; b++)
			{
				glm::vec3 normalB = d_normals[b];
				float distance = dotp(globalA, normalB);
				weightsum += sinc(distance);
			}

			d_weights[elements * interpindex + x] = 1.0f / weightsum;
		}
		else
		{
			d_weights[elements * interpindex + x] = 0.0f;
		}
	}
}

template <bool iscentered> __global__ void Exact3DWeightingKernel(tfloat* d_weights, int3 dims, glm::vec3* d_normals, int nimages, tfloat maxfreq)
{
	int idy = blockIdx.x;
	int idz = blockIdx.y;

	int x, y, z;
	if (!iscentered)
	{
		y = dims.y - 1 - ((idy + dims.y / 2 - 1) % dims.y);
		z = dims.z - 1 - ((idz + dims.z / 2 - 1) % dims.z);
	}
	else
	{
		y = idy;
		z = idz;
	}
	d_weights += (z * dims.y + y)  * (dims.x / 2 + 1);

	int elements = (dims.x / 2 + 1) * dims.y;
	glm::vec3 center = glm::vec3(dims.x / 2, dims.y / 2, dims.z / 2);

	for (int idx = threadIdx.x; idx < dims.x / 2 + 1; idx += blockDim.x)
	{
		if (!iscentered)
			x = dims.x / 2 - idx;
		else
			x = idx;

		glm::vec3 globalA = glm::vec3(idx, idy, idz) - center;
		if (glm::length(globalA) <= maxfreq)
		{
			float weightsum = 0.0f;

			for (int b = 0; b < nimages; b++)
			{
				glm::vec3 normalB = d_normals[b];
				float distance = dotp(globalA, normalB);
				weightsum += sinc(distance);
			}

			d_weights[x] = 1.0f / max(weightsum, 1.0f);
		}
		else
		{
			d_weights[x] = 0.0f;
		}
	}
}