#include "hip/hip_runtime.h"
#include "Prerequisites.cuh"
#include "Angles.cuh"
#include "DeviceFunctions.cuh"

texture<tfloat, 2> texBackprojImage;

#define SincWindow 16


////////////////////////////
//CUDA kernel declarations//
////////////////////////////

__global__ void ProjBackwardKernel(tfloat* d_volume, int3 dimsvolume, int3 dimsimage, glm::mat4 rotation, float weight);
__global__ void ProjBackwardSincKernel(tfloat* d_volume, int3 dimsvolume, tfloat* d_image, int2 dimsimage, glm::mat4 transform);


/////////////////////////////////////////////
//Equivalent of TOM's tom_backproj3d method//
/////////////////////////////////////////////

void d_ProjBackward(tfloat* d_volume, int3 dimsvolume, tfloat3 offsetfromcenter, tfloat* d_image, int3 dimsimage, tfloat3* h_angles, tfloat2* h_offsets, tfloat2* h_scales, T_INTERP_MODE mode, int batch)
{
	glm::mat4* h_transforms = (glm::mat4*)malloc(batch * sizeof(glm::mat4));
	for (int b = 0; b < batch; b++)
	{
		h_transforms[b] = Matrix4Translation(tfloat3(-dimsvolume.x / 2, -dimsvolume.y / 2, -dimsvolume.z / 2)) *
						  Matrix4Translation(offsetfromcenter) *
						  glm::transpose(Matrix4Euler(h_angles[b])) *
						  Matrix4Scale(tfloat3(1.0f / h_scales[b].x, 1.0f / h_scales[b].y, 1.0f)) *
						  Matrix4Translation(tfloat3(dimsimage.x / 2 - h_offsets[b].x, dimsimage.y / 2 - h_offsets[b].y, 0.0f));
	}

	if (mode == T_INTERP_LINEAR)
	{
		hipChannelFormatDesc descInput = hipCreateChannelDesc<tfloat>();
		texBackprojImage.normalized = false;
		texBackprojImage.filterMode = hipFilterModeLinear;

		size_t TpB = min(192, NextMultipleOf(dimsvolume.x, 32));
		dim3 grid = dim3((dimsvolume.x + TpB - 1) / TpB, dimsvolume.y, dimsvolume.z);
		for (int b = 0; b < batch; b++)
		{
			hipBindTexture2D(0,
				texBackprojImage,
				d_image + Elements(dimsimage) * b,
				descInput,
				dimsimage.x,
				dimsimage.y,
				dimsimage.x * sizeof(tfloat));


			hipUnbindTexture(texBackprojImage);
		}
	}
	else if (mode == T_INTERP_SINC)
	{
		dim3 TpB = dim3(SincWindow, SincWindow);
		dim3 grid = dim3(dimsvolume.x, dimsvolume.y, dimsvolume.z);

		for (int b = 0; b < batch; b++)
			ProjBackwardSincKernel <<<grid, TpB>>> (d_volume, dimsvolume, d_image + Elements(dimsimage) * b, toInt2(dimsimage.x, dimsimage.y), h_transforms[b]);
	}

	free(h_transforms);
}


////////////////
//CUDA kernels//
////////////////

__global__ void ProjBackwardKernel(tfloat* d_volume, int3 dimsvolume, int3 dimsimage, glm::mat4 rotation, float weight)
{
	int xvol = blockIdx.x * blockDim.x + threadIdx.x;
	if(xvol >= dimsvolume.x)
		return;

	glm::vec4 voxelpos = glm::vec4((tfloat)(xvol - dimsvolume.x / 2), 
								   (tfloat)((int)blockIdx.y - dimsvolume.y / 2), 
								   (tfloat)((int)blockIdx.z - dimsvolume.z / 2), 
								   1.0f);
	glm::vec4 rotated = rotation * voxelpos;

	rotated.x += (tfloat)(dimsimage.x / 2) + (tfloat)0.5;
	rotated.y += (tfloat)(dimsimage.y / 2) + (tfloat)0.5;
	d_volume[(blockIdx.z * dimsvolume.y + blockIdx.y) * dimsvolume.x + xvol] += weight * tex2D(texBackprojImage, 
																								rotated.x, 
																								rotated.y);
}

template <bool iscentered> __global__ void ProjBackwardSincKernel(tfloat* d_volume, int3 dimsvolume, tfloat* d_image, int2 dimsimage, glm::mat4 transform)
{
	__shared__ tfloat s_sums[SincWindow][SincWindow];
	s_sums[threadIdx.y][threadIdx.x] = 0.0f;

	int outx, outy, outz;
	if (!iscentered)
	{
		outx = gridDim.x / 2 - blockIdx.x;
		outy = gridDim.y - 1 - ((blockIdx.y + gridDim.y / 2 - 1) % gridDim.y);
		outz = gridDim.z - 1 - ((blockIdx.z + gridDim.z / 2 - 1) % gridDim.z);
	}
	else
	{
		outx = blockIdx.x;
		outy = blockIdx.y;
		outz = blockIdx.z;
	}

	glm::vec4 position = glm::vec4(blockIdx.x, blockIdx.y, blockIdx.z, 1.0f);
	position = transform * position;
	if (position.x < 0 || position.x > dimsimage.x - 1 || position.y < 0 || position.y > dimsimage.y - 1)
	{
		d_volume[(outz * gridDim.y + outy) * gridDim.x + outx] = 0.0f;
		return;
	}

	short startx = (short)position.x - SincWindow / 2;
	short starty = (short)position.y - SincWindow / 2;
	float sum = 0.0f;

	for (short y = threadIdx.y; y < SincWindow; y += blockDim.y)
	{
		short yy = y + starty;
		float weighty = sinc(position.y - (float)yy);
		int addressy = (yy + dimsimage.y) % dimsimage.y;

		for (int x = threadIdx.x; x < SincWindow; x += blockDim.x)
		{
			int xx = x + startx;
			float weight = sinc(position.x - (float)xx) * weighty;
			int addressx = (xx + dimsimage.x) % dimsimage.x;

			sum += d_image[addressy * dimsimage.x + addressx] * weight;
		}
	}
	s_sums[threadIdx.y][threadIdx.x] = sum;
	__syncthreads();
	
	if (threadIdx.x == 0)
	{
		#pragma unroll
		for (char i = 1; i < SincWindow; i++)
			sum += s_sums[threadIdx.y][i];
		s_sums[threadIdx.y][0] = sum;
	}
	__syncthreads();

	if (threadIdx.y == 0 && threadIdx.x == 0)
	{
		#pragma unroll
		for (char i = 1; i < SincWindow; i++)
			sum += s_sums[i][0];
		d_volume[(outz * gridDim.y + outy) * gridDim.x + outx] = sum;
	}
}