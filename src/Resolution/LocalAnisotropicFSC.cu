#include "hip/hip_runtime.h"
#include "Prerequisites.cuh"
#include "FFT.cuh"
#include "Helper.cuh"
#include "Generics.cuh"
#include "Masking.cuh"
#include "Resolution.cuh"


///////////////////////////
//CUDA kernel declaration//
///////////////////////////



///////////////////////////////////////////////
//Local Anisotropic Fourier Shell Correlation//
///////////////////////////////////////////////

void d_LocalAnisotropicFSC(tfloat* d_volume1, tfloat* d_volume2, int3 dimsvolume, tfloat* d_resolution, int windowsize, int maxradius, int2 anglesteps, tfloat threshold)
{
	//maxradius *= 2;

	//tfloat* h_volume1 = (tfloat*)MallocPinnedFromDeviceArray(*d_volume1, Elements(dimsvolume) * sizeof(tfloat));
	//tfloat* h_volume2 = (tfloat*)MallocPinnedFromDeviceArray(*d_volume2, Elements(dimsvolume) * sizeof(tfloat));

	int samplesperelement = anglesteps.x * anglesteps.y;

	int3 dimspaddedvolume = toInt3(dimsvolume.x + windowsize, dimsvolume.y + windowsize, dimsvolume.z + windowsize);
	int3 dimswindow = toInt3(windowsize, windowsize, windowsize);
	int3 dimspaddedwindow = toInt3(windowsize * 2, windowsize * 2, windowsize * 2);

	tfloat* d_paddedvolume1;
	hipMalloc((void**)&d_paddedvolume1, Elements(dimspaddedvolume) * sizeof(tfloat));
	d_Pad(d_volume1, d_paddedvolume1, dimsvolume, dimspaddedvolume, T_PAD_MIRROR, (tfloat)0);
	//hipFree(*d_volume1);
	tfloat* d_paddedvolume2;
	hipMalloc((void**)&d_paddedvolume2, Elements(dimspaddedvolume) * sizeof(tfloat));
	d_Pad(d_volume2, d_paddedvolume2, dimsvolume, dimspaddedvolume, T_PAD_MIRROR, (tfloat)0);
	//hipFree(*d_volume2);

	uint batchmemory = 256 * 1024 * 1024;
	uint windowmemory = Elements(dimswindow) * sizeof(tfloat);
	uint batchsize = batchmemory / windowmemory;

	tfloat *d_extracts1, *d_extracts2;
	hipMalloc((void**)&d_extracts1, Elements(dimswindow) * batchsize * sizeof(tfloat));
	hipMalloc((void**)&d_extracts2, Elements(dimswindow) * batchsize * sizeof(tfloat));
	/*tfloat *d_paddedextracts1, *d_paddedextracts2;
	hipMalloc((void**)&d_paddedextracts1, Elements(dimspaddedwindow) * batchsize * sizeof(tfloat));
	hipMalloc((void**)&d_paddedextracts2, Elements(dimspaddedwindow) * batchsize * sizeof(tfloat));*/

	tfloat* d_mask = CudaMallocValueFilled(Elements(dimswindow) * batchsize, (tfloat)1);
	d_HannMask(d_mask, d_mask, dimswindow, NULL, NULL, batchsize);
	//tfloat* h_mask = (tfloat*)MallocFromDeviceArray(d_mask, Elements(dimswindow) * batchsize * sizeof(tfloat));
	//free(h_mask);

	int3* h_extractcenters;
	hipHostMalloc((void**)&h_extractcenters, batchsize * sizeof(int3));
	int3* d_extractcenters;
	hipMalloc((void**)&d_extractcenters, batchsize * sizeof(int3));

	tfloat* d_fsccurves;
	hipMalloc((void**)&d_fsccurves, maxradius * samplesperelement * batchsize * sizeof(tfloat));
	tfloat* d_resvalues;
	hipMalloc((void**)&d_resvalues, batchsize * samplesperelement * sizeof(tfloat));

	hipfftHandle planforw = d_FFTR2CGetPlan(DimensionCount(dimswindow), dimswindow, batchsize);

	int3 dimstrimmed = toInt3(dimsvolume.x - windowsize, dimsvolume.y - windowsize, dimsvolume.z - windowsize);

	int elements = Elements(dimsvolume);
	int elementsxy = dimsvolume.x * dimsvolume.y;
	int elementswindow = Elements(dimswindow);


	for (int i = 0; i < elements; i += batchsize)
	{
		for(int b = i; b < min(elements, i + batchsize); b++)
		{
			int z = b / elementsxy;
			int y = (b - z * elementsxy) / dimsvolume.x;
			int x = b % dimsvolume.x;

			h_extractcenters[b - i] = toInt3(x, y, z);
		}
		hipMemcpy(d_extractcenters, h_extractcenters, batchsize * sizeof(int3), hipMemcpyHostToDevice);

		d_ExtractMany(d_paddedvolume1, d_extracts1, dimspaddedvolume, dimswindow, d_extractcenters, batchsize);
		d_ExtractMany(d_paddedvolume2, d_extracts2, dimspaddedvolume, dimswindow, d_extractcenters, batchsize);

		d_MultiplyByVector(d_extracts1, d_mask, d_extracts1, elementswindow, batchsize);
		d_MultiplyByVector(d_extracts2, d_mask, d_extracts2, elementswindow, batchsize);
		
		d_AnisotropicFSCMap(d_extracts1, d_extracts2, dimswindow, d_fsccurves, anglesteps, maxradius, T_FSC_MODE::T_FSC_THRESHOLD, threshold, &planforw, batchsize);

		d_ValueFill(d_resvalues, batchsize * samplesperelement, (tfloat)-1);
		d_FirstIndexOf(d_fsccurves, d_resvalues, maxradius, threshold, T_INTERP_LINEAR, batchsize * samplesperelement);
		/*tfloat* h_resvalues = (tfloat*)MallocFromDeviceArray(d_resvalues, batchsize * samplesperelement * sizeof(tfloat));
		free(h_resvalues);*/

		hipMemcpy(d_resolution + i * samplesperelement, d_resvalues, min(batchsize, elements - i) * samplesperelement * sizeof(tfloat), hipMemcpyDeviceToDevice);

		//break;
		printf("%f\n", (tfloat)i / (tfloat)elements * (tfloat)100);
	}


	hipfftDestroy(planforw);
	hipFree(d_resvalues);
	hipHostFree(d_fsccurves);
	hipFree(d_extractcenters);
	hipHostFree(h_extractcenters);
	hipFree(d_mask);
	/*hipFree(d_paddedextracts2);
	hipFree(d_paddedextracts1);*/
	hipFree(d_extracts2);
	hipFree(d_extracts1);
	hipFree(d_paddedvolume1);
	hipFree(d_paddedvolume2);
}