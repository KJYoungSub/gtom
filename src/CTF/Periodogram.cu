#include "hip/hip_runtime.h"
#include "Prerequisites.cuh"
#include "Angles.cuh"
#include "CTF.cuh"
#include "CubicInterp.cuh"
#include "DeviceFunctions.cuh"
#include "FFT.cuh"
#include "Generics.cuh"
#include "Helper.cuh"
#include "ImageManipulation.cuh"
#include "Masking.cuh"
#include "Transformation.cuh"


namespace gtom
{
	//////////////////////////////////////////////////////
	//Calculate power spectrum based on multiple regions//
	//////////////////////////////////////////////////////

	void d_CTFPeriodogram(tfloat* d_image, int2 dimsimage, float overlapfraction, int2 dimsregion, int2 dimspadded, tfloat* d_output2d, bool dopost)
	{
		// Create uniform grid over the image
		int2 regions;
		int3* h_origins = GetEqualGridSpacing(dimsimage, dimsregion, overlapfraction, regions);
		int3* d_origins = (int3*)CudaMallocFromHostArray(h_origins, Elements2(regions) * sizeof(int3));
		free(h_origins);

		int norigins = Elements2(regions);

		tfloat* d_temp2d;
		hipMalloc((void**)&d_temp2d, ElementsFFT2(dimspadded) * norigins * sizeof(tfloat));

		// Call the custom-grid version to extract 2D spectra
		d_CTFPeriodogram(d_image, dimsimage, d_origins, norigins, dimsregion, dimspadded, d_temp2d, dopost);

		d_ReduceMean(d_temp2d, d_output2d, ElementsFFT2(dimspadded), norigins);

		hipFree(d_temp2d);
		hipFree(d_origins);
	}

	void d_CTFPeriodogram(tfloat* d_image, int2 dimsimage, int3* d_origins, int norigins, int2 dimsregion, int2 dimspadded, tfloat* d_output2d, bool dopost)
	{
		int memlimit = 128 << 20;
		int batchsize = tmin(norigins, memlimit / (int)(Elements2(dimsregion) * 2 * sizeof(tfloat)));

		tfloat* d_extracted;
		hipMalloc((void**)&d_extracted, batchsize * Elements2(dimspadded) * sizeof(tfloat));
		tcomplex* d_extractedft;
		hipMalloc((void**)&d_extractedft, batchsize * ElementsFFT2(dimspadded) * sizeof(tcomplex));

		for (int b = 0; b < norigins; b += batchsize)
		{
			int curbatch = tmin(batchsize, norigins - b);

			d_ExtractMany(d_image, d_extracted, toInt3(dimsimage), toInt3(dimsregion), d_origins + b, curbatch);
			//d_WriteMRC(d_extracted, toInt3(dimsregion.x, dimsregion.y, curbatch), "d_extracted.mrc");

			d_NormMonolithic(d_extracted, d_extracted, Elements2(dimsregion), T_NORM_MEAN01STD, curbatch);
			d_HammingMask(d_extracted, d_extracted, toInt3(dimsregion), NULL, NULL, curbatch);
			//d_HammingMaskBorderDistance(d_extracted, d_extracted, toInt3(dimsregion), dimsregion.x / 4, curbatch);
			if (dimsregion.x != dimspadded.x || dimsregion.y != dimspadded.y)
			{
				d_Pad(d_extracted, (tfloat*)d_extractedft, toInt3(dimsregion), toInt3(dimspadded), T_PAD_VALUE, (tfloat)0, curbatch);
				d_NormMonolithic((tfloat*)d_extractedft, d_extracted, Elements2(dimspadded), T_NORM_MEAN01STD, curbatch);
			}
			else
			{
				d_NormMonolithic(d_extracted, d_extracted, Elements2(dimspadded), T_NORM_MEAN01STD, curbatch);
			}
			//d_WriteMRC(d_extracted, toInt3(dimspadded.x, dimspadded.y, curbatch), "d_extracted.mrc");
			d_FFTR2C(d_extracted, d_extractedft, 2, toInt3(dimspadded), curbatch);
			d_Abs(d_extractedft, d_extracted, curbatch * ElementsFFT2(dimspadded));
			//d_WriteMRC(d_extracted, toInt3(dimspadded.x / 2 + 1, dimspadded.y, curbatch), "d_extractedft.mrc");

			if (dopost)
			{
				d_AddScalar(d_extracted, d_extracted, curbatch * ElementsFFT2(dimspadded), (tfloat)1e-6);
				d_Log(d_extracted, d_extracted, curbatch * ElementsFFT2(dimspadded));
				d_MultiplyByVector(d_extracted, d_extracted, d_extracted, ElementsFFT2(dimspadded) * curbatch);
			}

			d_RemapHalfFFT2Half(d_extracted, d_output2d + b * ElementsFFT2(dimspadded), toInt3(dimspadded), curbatch);
			//d_WriteMRC(d_output2d + b * ElementsFFT2(dimspadded), toInt3(dimspadded.x / 2 + 1, dimspadded.y, curbatch), "d_extractedoutput.mrc");
		}

		hipFree(d_extractedft);
		hipFree(d_extracted);
	}
}