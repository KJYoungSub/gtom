#include "hip/hip_runtime.h"
#include "Prerequisites.cuh"


namespace gtom
{
	////////////////////////////
	//CUDA kernel declarations//
	////////////////////////////

	template <class T> __global__ void SphereMaskKernel(T* d_input, T* d_output, int3 size, tfloat radius, tfloat sigma, tfloat3 center);
	__global__ void SphereMaskFTKernel(tfloat* d_input, tfloat* d_output, int3 dims, int radius2);


	////////////////
	//Host methods//
	////////////////

	template <class T> void d_SphereMask(T* d_input,
										T* d_output,
										int3 size,
										tfloat* radius,
										tfloat sigma,
										tfloat3* center,
										int batch)
	{
		tfloat _radius = radius != NULL ? *radius : min(min(size.x, size.y), size.z > 1 ? size.z : size.x) / 2;
		tfloat3 _center = center != NULL ? *center : tfloat3(size.x / 2, size.y / 2, size.z / 2);

		int TpB = 256;
		dim3 grid = dim3(size.y, size.z, batch);
		SphereMaskKernel<T> << <grid, TpB >> > (d_input, d_output, size, _radius, sigma, _center);
	}
	template void d_SphereMask<tfloat>(tfloat* d_input, tfloat* d_output, int3 size, tfloat* radius, tfloat sigma, tfloat3* center, int batch);
	template void d_SphereMask<tcomplex>(tcomplex* d_input, tcomplex* d_output, int3 size, tfloat* radius, tfloat sigma, tfloat3* center, int batch);

	void d_SphereMaskFT(tfloat* d_input, tfloat* d_output, int3 dims, int radius, uint batch)
	{
		int TpB = tmin(128, NextMultipleOf(dims.x, 32));
		dim3 grid = dim3(dims.y, dims.z, batch);
		SphereMaskFTKernel <<<grid, TpB>>> (d_input, d_output, dims, radius * radius);
	}


	////////////////
	//CUDA kernels//
	////////////////

	template <class T> __global__ void SphereMaskKernel(T* d_input, T* d_output, int3 size, tfloat radius, tfloat sigma, tfloat3 center)
	{
		if (threadIdx.x >= size.x)
			return;

		//For batch mode
		int offset = blockIdx.z * size.x * size.y * size.z + blockIdx.y * size.x * size.y + blockIdx.x * size.x;

		tfloat xsq, ysq, zsq, length;
		T maskvalue;

		//Squared y and z distance from center
		ysq = (tfloat)blockIdx.x - center.y;
		ysq *= ysq;
		if (size.z > 1)
		{
			zsq = (tfloat)blockIdx.y - center.z;
			zsq *= zsq;
		}
		else
			zsq = 0;

		for (int x = threadIdx.x; x < size.x; x += blockDim.x)
		{
			xsq = (tfloat)x - center.x;
			xsq *= xsq;
			//Distance from center
			length = sqrt(xsq + ysq + zsq);

			if (length < radius)
				maskvalue = 1;
			else
			{
				//Smooth border
				if (sigma > (tfloat)0)
				{
					maskvalue = tmax(0, (cos(tmin(1.0f, (length - radius) / sigma) * PI) + 1.0f) * 0.5f);
				}
				//Hard border
				else
					maskvalue = 0;
			}

			//Write masked input to output
			d_output[offset + x] = maskvalue * d_input[offset + x];
		}
	}

	template<> __global__ void SphereMaskKernel<tcomplex>(tcomplex* d_input, tcomplex* d_output, int3 size, tfloat radius, tfloat sigma, tfloat3 center)
	{
		if (threadIdx.x >= size.x)
			return;

		//For batch mode
		int offset = blockIdx.z * size.x * size.y * size.z + blockIdx.y * size.x * size.y + blockIdx.x * size.x;

		tfloat xsq, ysq, zsq, length;
		tfloat maskvalue;

		//Squared y and z distance from center
		ysq = (tfloat)(blockIdx.x + 1) - center.y;
		ysq *= ysq;
		if (size.z > 1)
		{
			zsq = (tfloat)(blockIdx.y + 1) - center.z;
			zsq *= zsq;
		}
		else
			zsq = 0;

		for (int x = threadIdx.x; x < size.x; x += blockDim.x)
		{
			xsq = (tfloat)(x + 1) - center.x;
			xsq *= xsq;
			//Distance from center
			length = sqrt(xsq + ysq + zsq);

			if (length < radius)
				maskvalue = 1;
			else
			{
				//Smooth border
				if (sigma > (tfloat)0)
				{
					maskvalue = (cos(tmin(1.0f, (length - radius) / sigma) * PI) + 1.0f) * 0.5f;
					if (maskvalue < (tfloat)0.1353)
						maskvalue = 0;
				}
				//Hard border
				else
					maskvalue = 0;
			}

			//Write masked input to output
			d_output[offset + x].x = maskvalue * d_input[offset + x].x;
			d_output[offset + x].y = maskvalue * d_input[offset + x].y;
		}
	}

	__global__ void SphereMaskFTKernel(tfloat* d_input, tfloat* d_output, int3 dims, int radius2)
	{
		int z = blockIdx.y;
		int y = blockIdx.x;

		d_input +=  blockIdx.z * ElementsFFT(dims) + (z * dims.y + y) * (dims.x / 2 + 1);
		d_output += blockIdx.z * ElementsFFT(dims) + (z * dims.y + y) * (dims.x / 2 + 1);

		int zp = z < dims.z / 2 + 1 ? z : z - dims.x;
		zp *= zp;
		int yp = y < dims.y / 2 + 1 ? y : y - dims.x;
		yp *= yp;

		for (int x = threadIdx.x; x < dims.x / 2 + 1; x += blockDim.x)
		{
			int r = x * x + yp + zp;

			if (r < radius2)
				d_output[x] = d_input[x];
			else
				d_output[x] = 0;
		}
	}
}