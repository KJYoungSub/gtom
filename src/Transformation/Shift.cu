#include "hip/hip_runtime.h"
#include "Prerequisites.cuh"
#include "FFT.cuh"


////////////////////////////
//CUDA kernel declarations//
////////////////////////////

template<int ndims, bool iszerocentered> __global__ void ShiftFourierKernel(tcomplex* d_input, tcomplex* d_output, int3 dims, tfloat3 delta);
__global__ void ShiftIntegerKernel(tfloat* d_input, tfloat* d_output, int3 dims, int3 delta);


////////////////////////////////////////
//Equivalent of TOM's tom_shift method//
////////////////////////////////////////

void d_Shift(tfloat* d_input, tfloat* d_output, int3 dims, tfloat3* delta, hipfftHandle* planforw, hipfftHandle* planback, tcomplex* d_sharedintermediate, int batch)
{
	tcomplex* d_intermediate = NULL;
	if(d_sharedintermediate == NULL)
		CudaSafeCall(hipMalloc((void**)&d_intermediate, ElementsFFT(dims) * sizeof(tcomplex)));
	else
		d_intermediate = d_sharedintermediate;

	for (int b = 0; b < batch; b++)
	{
		if(fmod(delta[b].x, (tfloat)1) != (tfloat)0 || fmod(delta[b].y, (tfloat)1) != (tfloat)0 || fmod(delta[b].z, (tfloat)1) != (tfloat)0)
		{
			tfloat3 normdelta = tfloat3(delta[b].x / (tfloat)dims.x, delta[b].y / (tfloat)dims.y, delta[b].z / (tfloat)dims.z);

			if(planforw == NULL)
				d_FFTR2C(d_input + Elements(dims) * b, d_intermediate, DimensionCount(dims), dims);
			else
				d_FFTR2C(d_input + Elements(dims) * b, d_intermediate, planforw);

			int TpB = min(256, NextMultipleOf(dims.x / 2 + 1, 32));
			dim3 grid = dim3(((dims.x / 2 + 1) + TpB - 1) / TpB, dims.y, dims.z);
			if(DimensionCount(dims) == 3)
				ShiftFourierKernel <3, false> <<<grid, TpB>>> (d_intermediate, d_intermediate, dims, normdelta);
			else
				ShiftFourierKernel <2, false> <<<grid, TpB>>> (d_intermediate, d_intermediate, dims, normdelta);
			hipStreamQuery(0);
			
			if(planback == NULL)
				d_IFFTC2R(d_intermediate, d_output + Elements(dims) * b, DimensionCount(dims), dims);
			else
				d_IFFTC2R(d_intermediate, d_output + Elements(dims) * b, planback, dims);
		}
		else
		{
			int TpB = min(256, NextMultipleOf(dims.x, 32));
			dim3 grid = dim3((dims.x + TpB - 1) / TpB, dims.y, dims.z);
			ShiftIntegerKernel <<<grid, TpB>>> (d_input + Elements(dims) * b, d_input == d_output ? (tfloat*)d_intermediate : (d_output + Elements(dims) * b), dims, toInt3((int)delta[b].x, (int)delta[b].y, (int)delta[b].z));
			hipStreamQuery(0);

			if(d_input == d_output)
				hipMemcpy(d_output + Elements(dims) * b, d_intermediate, Elements(dims) * sizeof(tfloat), hipMemcpyDeviceToDevice);
		}
	}

	if(d_sharedintermediate == NULL)
		hipFree(d_intermediate);
}

void d_Shift(tcomplex* d_input, tcomplex* d_output, int3 dims, tfloat3* delta, bool iszerocentered, int batch)
{
	for (int b = 0; b < batch; b++)
	{
		tfloat3 normdelta = tfloat3(delta[b].x / (tfloat)dims.x, delta[b].y / (tfloat)dims.y, delta[b].z / (tfloat)dims.z);

		int TpB = min(256, NextMultipleOf(dims.x / 2 + 1, 32));
		dim3 grid = dim3(((dims.x / 2 + 1) + TpB - 1) / TpB, dims.y, dims.z);
		if(!iszerocentered)
		{
			if(DimensionCount(dims) == 3)
				ShiftFourierKernel <3, false> <<<grid, TpB>>> (d_input + ElementsFFT(dims) * b, d_output + ElementsFFT(dims) * b, dims, normdelta);
			else
				ShiftFourierKernel <2, false> <<<grid, TpB>>> (d_input + ElementsFFT(dims) * b, d_output + ElementsFFT(dims) * b, dims, normdelta);
		}
		else
		{
			if(DimensionCount(dims) == 3)
				ShiftFourierKernel <3, true> <<<grid, TpB>>> (d_input + ElementsFFT(dims) * b, d_output + ElementsFFT(dims) * b, dims, normdelta);
			else
				ShiftFourierKernel <2, true> <<<grid, TpB>>> (d_input + ElementsFFT(dims) * b, d_output + ElementsFFT(dims) * b, dims, normdelta);
		}
		hipStreamQuery(0);
	}
}


////////////////
//CUDA kernels//
////////////////

template<int ndims, bool iszerocentered> __global__ void ShiftFourierKernel(tcomplex* d_input, tcomplex* d_output, int3 dims, tfloat3 delta)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if(idx > dims.x / 2)
		return;
	int idy = blockIdx.y;
	int idz = blockIdx.z;

	int x, y, z;
	if(!iszerocentered)
	{
		x = idx;
		y = ((idy + ((dims.y + 1) / 2)) % dims.y) - dims.y / 2;
		z = ((idz + ((dims.z + 1) / 2)) % dims.z) - dims.z / 2;
	}
	else
	{
		x = dims.x / 2 - idx;
		y = dims.y / 2 - idy;
		z = dims.z / 2 - idz;
	}

	tfloat factor = (delta.x * (tfloat)x + delta.y * (tfloat)y + (ndims > 2 ? delta.z * (tfloat)z : (tfloat)0)) * (tfloat)PI2;
	tcomplex multiplicator = make_hipComplex(cos(factor), sin(-factor));

	size_t id = (idz * dims.y + idy) * (dims.x / 2 + 1) + idx;
	d_output[id] = cmul(d_input[id], multiplicator);
}

__global__ void ShiftIntegerKernel(tfloat* d_input, tfloat* d_output, int3 dims, int3 delta)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	if(x >= dims.x)
		return;

	d_output[(((dims.z + (int)blockIdx.z + delta.z) % dims.z) * dims.y + ((dims.y + (int)blockIdx.y + delta.y) % dims.y)) * dims.x + ((dims.x + x + delta.x) % dims.x)] = d_input[(blockIdx.z * dims.y + blockIdx.y) * dims.x + x];
}