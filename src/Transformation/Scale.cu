#include "hip/hip_runtime.h"
#include "Prerequisites.cuh"
#include "CubicInterp.cuh"
#include "FFT.cuh"
#include "Generics.cuh"
#include "Helper.cuh"


namespace gtom
{
	////////////////////////////
	//CUDA kernel declarations//
	////////////////////////////

	template <int ndims, bool cubicinterp> __global__ void InterpolateKernel(tfloat* d_output, int3 dimsnew, cudaTex t_input, int3 dimsold, tfloat3 factor, tfloat3 offset);


	////////////////////////////////////////////////////////////////////////////////
	//Combines the functionality of TOM's tom_rescale and MATLAB's interp* methods// 
	////////////////////////////////////////////////////////////////////////////////

	void d_Scale(tfloat* d_input, tfloat* d_output, int3 olddims, int3 newdims, T_INTERP_MODE mode, hipfftHandle* planforw, hipfftHandle* planback, int batch)
	{
		//Both sizes should have an equal number of dimensions
		int ndims = DimensionCount(olddims);
		if (ndims != DimensionCount(newdims))
			throw;

		//All new dimensions must be either bigger than the old or smaller, not mixed
		int biggerdims = 0;
		for (int i = 0; i < ndims; i++)
			if (((int*)&newdims)[i] >= ((int*)&olddims)[i])
				biggerdims++;
		if (biggerdims != 0 && biggerdims != ndims)
			throw;

		if (mode == T_INTERP_LINEAR || mode == T_INTERP_CUBIC)
		{
			hipArray* a_image;
			cudaTex t_image;
			tfloat* d_temp;
			hipMalloc((void**)&d_temp, Elements(olddims) * sizeof(tfloat));

			for (int b = 0; b < batch; b++)
			{
				hipMemcpy(d_temp, d_input + Elements(olddims) * b, Elements(olddims) * sizeof(tfloat), hipMemcpyDeviceToDevice);
				if (mode == T_INTERP_CUBIC)
				{
					if (ndims == 2)
						d_CubicBSplinePrefilter2D(d_temp, toInt2(olddims));
					else if (ndims == 3)
						d_CubicBSplinePrefilter3D(d_temp, olddims);
				}
				if (ndims == 3)
					d_BindTextureTo3DArray(d_temp, a_image, t_image, olddims, hipFilterModeLinear, false);
				else
					d_BindTextureToArray(d_temp, a_image, t_image, toInt2(olddims), hipFilterModeLinear, false);

				dim3 TpB, grid;
				if (ndims > 1)
				{
					TpB = dim3(16, 16);
					grid = dim3((newdims.x + 15) / 16, (newdims.y + 15) / 16, newdims.z);
				}
				else
				{
					TpB = dim3(256);
					grid = dim3((newdims.x + 255) / 256);
				}

				tfloat3 factor = tfloat3((tfloat)olddims.x / (tfloat)newdims.x, (tfloat)olddims.y / (tfloat)newdims.y, (tfloat)olddims.z / (tfloat)newdims.z);
				tfloat3 offset = tfloat3(0.5f * factor.x, 0.5f * factor.y, 0.5f * factor.z);

				if (mode == T_INTERP_CUBIC)
				{
					if (ndims == 1)
						InterpolateKernel<1, true> << <grid, TpB >> > (d_output + Elements(newdims) * b, newdims, t_image, olddims, factor, offset);
					else if (ndims == 2)
						InterpolateKernel<2, true> << <grid, TpB >> > (d_output + Elements(newdims) * b, newdims, t_image, olddims, factor, offset);
					else if (ndims == 3)
						InterpolateKernel<3, true> << <grid, TpB >> > (d_output + Elements(newdims) * b, newdims, t_image, olddims, factor, offset);
				}
				else
				{
					if (ndims == 1)
						InterpolateKernel<1, false> << <grid, TpB >> > (d_output + Elements(newdims) * b, newdims, t_image, olddims, factor, offset);
					else if (ndims == 2)
						InterpolateKernel<2, false> << <grid, TpB >> > (d_output + Elements(newdims) * b, newdims, t_image, olddims, factor, offset);
					else if (ndims == 3)
						InterpolateKernel<3, false> << <grid, TpB >> > (d_output + Elements(newdims) * b, newdims, t_image, olddims, factor, offset);
				}

				hipDestroyTextureObject(t_image);
				hipFreeArray(a_image);
			}

			hipFree(d_temp);
		}
		else if (mode == T_INTERP_FOURIER)
		{
			tcomplex* d_inputFFT;
			hipMalloc((void**)&d_inputFFT, ElementsFFT(olddims) * batch * sizeof(tcomplex));
			tcomplex* d_outputFFT;
			hipMalloc((void**)&d_outputFFT, ElementsFFT(newdims) * batch * sizeof(tcomplex));

			tfloat normfactor = (tfloat)1 / (tfloat)Elements(olddims);

			if (planforw == NULL)
				d_FFTR2C(d_input, d_inputFFT, ndims, olddims, batch);
			else
				d_FFTR2C(d_input, d_inputFFT, planforw);

			if (newdims.x > olddims.x)
				d_FFTPad(d_inputFFT, d_outputFFT, olddims, newdims, batch);
			else
				d_FFTCrop(d_inputFFT, d_outputFFT, olddims, newdims, batch);

			if (planback == NULL)
				d_IFFTC2R(d_outputFFT, d_output, ndims, newdims, batch, false);
			else
				d_IFFTC2R(d_outputFFT, d_output, planback);

			d_MultiplyByScalar(d_output, d_output, Elements(newdims) * batch, normfactor);

			hipFree(d_inputFFT);
			hipFree(d_outputFFT);
		}
	}


	////////////////
	//CUDA kernels//
	////////////////

	template <int ndims, bool cubicinterp> __global__ void InterpolateKernel(tfloat* d_output, int3 dimsnew, cudaTex t_input, int3 dimsold, tfloat3 factor, tfloat3 offset)
	{
		int idx = blockIdx.x * blockDim.x + threadIdx.x;
		if (idx >= dimsnew.x)
			return;
		int idy = blockIdx.y * blockDim.y + threadIdx.y;
		if (idy >= dimsnew.y)
			return;
		int idz = blockIdx.z;

		tfloat3 position = tfloat3(idx - dimsnew.x / 2, idy - dimsnew.y / 2, idz - dimsnew.z / 2);
		position = tfloat3(position.x * factor.x, position.y * factor.y, position.z * factor.z);
		position = tfloat3(position.x + dimsold.x / 2 + offset.x, position.y + dimsold.y / 2 + offset.y, position.z + dimsold.z / 2 + offset.z);

		if (cubicinterp)
		{
			if (ndims == 1)
				d_output[idx] = cubicTex1D(t_input, position.x);
			else if (ndims == 2)
				d_output[idy * dimsnew.x + idx] = cubicTex2D(t_input, position.x, position.y);
			else if (ndims == 3)
				d_output[(idz * dimsnew.y + idy) * dimsnew.x + idx] = cubicTex3D(t_input, position.x, position.y, position.z);
		}
		else
		{
			if (ndims == 1)
				d_output[idx] = tex1D<tfloat>(t_input, position.x);
			else if (ndims == 2)
				d_output[idy * dimsnew.x + idx] = tex2D<tfloat>(t_input, position.x, position.y);
			else if (ndims == 3)
				d_output[(idz * dimsnew.y + idy) * dimsnew.x + idx] = tex3D<tfloat>(t_input, position.x, position.y, position.z);
		}
	}
}