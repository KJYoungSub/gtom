#include "hip/hip_runtime.h"
#include "Prerequisites.cuh"
#include "Angles.cuh"
#include "CubicInterp.cuh"
#include "DeviceFunctions.cuh"
#include "FFT.cuh"
#include "Generics.cuh"
#include "Helper.cuh"
#include "Transformation.cuh"


////////////////////////////
//CUDA kernel declarations//
////////////////////////////

template<bool cubicinterp, bool zerocentered> __global__ void Rotate3DKernel(cudaTex t_input, tfloat* d_output, int3 dims, glm::mat4 transform);
template<bool cubicinterp, bool outputzerocentered> __global__ void Rotate2DFTKernel(cudaTex t_Re, cudaTex t_Im, tcomplex* d_output, int3 dims, glm::mat2 transform, tfloat maxfreq);
template<bool cubicinterp, bool zerocentered> __global__ void Rotate3DFTKernel(cudaTex t_Re, cudaTex t_Im, tcomplex* d_output, int3 dims, glm::mat4* d_transform, float maxfreq2);
template<bool cubicinterp, bool zerocentered> __global__ void Rotate3DFTKernel(cudaTex t_Re, tfloat* d_output, int3 dims, glm::mat4* d_transform, float maxfreq2);


////////////////////
//Rotate 3D volume//
////////////////////

void d_Rotate3D(tfloat* d_volume, tfloat* d_output, int3 dims, tfloat3* h_angles, int nangles, T_INTERP_MODE mode, bool iszerocentered)
{
	tfloat* d_temp;
	if (mode == T_INTERP_CUBIC)
		hipMalloc((void**)&d_temp, Elements(dims) * sizeof(tfloat));

	hipArray* a_input;
	cudaTex t_input;
	if (mode == T_INTERP_LINEAR)
		d_BindTextureTo3DArray(d_volume, a_input, t_input, dims, hipFilterModeLinear, false);
	else
	{
		hipMemcpy(d_temp, d_volume, Elements(dims) * sizeof(tfloat), hipMemcpyDeviceToDevice);
		d_CubicBSplinePrefilter3D(d_temp, dims);
		d_BindTextureTo3DArray(d_temp, a_input, t_input, dims, hipFilterModeLinear, false);
	}

	for (int b = 0; b < nangles; b++)
	{
		glm::mat4 transform = Matrix4Translation(tfloat3(dims.x / 2 + 0.5f, dims.y / 2 + 0.5f, dims.z / 2 + 0.5f)) *
							  glm::transpose(Matrix4Euler(h_angles[b])) *
							  Matrix4Translation(tfloat3(-dims.x / 2, -dims.y / 2, -dims.z / 2));

		dim3 TpB = dim3(16, 16);
		dim3 grid = dim3((dims.x + 15) / 16, (dims.y + 15) / 16, dims.z);

		if (iszerocentered)
		{
			if (mode == T_INTERP_LINEAR)
				Rotate3DKernel<false, true> << <grid, TpB >> > (t_input, d_output, dims, transform);
			else if (mode == T_INTERP_CUBIC)
				Rotate3DKernel<true, true> << <grid, TpB >> > (t_input, d_output, dims, transform);
		}
		else
		{
			if (mode == T_INTERP_LINEAR)
				Rotate3DKernel<false, false> << <grid, TpB >> > (t_input, d_output, dims, transform);
			else if (mode == T_INTERP_CUBIC)
				Rotate3DKernel<true, false> << <grid, TpB >> > (t_input, d_output, dims, transform);
		}
	}

	hipDestroyTextureObject(t_input);
	hipFreeArray(a_input);

	if (mode == T_INTERP_CUBIC)
		hipFree(d_temp);
}


//////////////////////////////
//Rotate 2D in Fourier space//
//////////////////////////////

void d_Rotate2DFT(tcomplex* d_input, tcomplex* d_output, int3 dims, tfloat* angles, tfloat maxfreq, T_INTERP_MODE mode, bool isoutputzerocentered, int batch)
{
	tfloat* d_real;
	hipMalloc((void**)&d_real, ElementsFFT(dims) * sizeof(tfloat));
	tfloat* d_imag;
	hipMalloc((void**)&d_imag, ElementsFFT(dims) * sizeof(tfloat));

	for (int b = 0; b < batch; b++)
	{
		d_ConvertTComplexToSplitComplex(d_input + ElementsFFT(dims) * b, d_real, d_imag, ElementsFFT(dims));

		if(mode == T_INTERP_CUBIC)
		{
			d_CubicBSplinePrefilter2D(d_real, toInt2(dims.x / 2 + 1, dims.y));
			d_CubicBSplinePrefilter2D(d_imag, toInt2(dims.x / 2 + 1, dims.y));
		}

		hipArray* a_Re;
		hipArray* a_Im;
		cudaTex t_Re, t_Im;
		d_BindTextureToArray(d_real, a_Re, t_Re, toInt2(dims.x / 2 + 1, dims.y), hipFilterModeLinear, false);
		d_BindTextureToArray(d_imag, a_Im, t_Im, toInt2(dims.x / 2 + 1, dims.y), hipFilterModeLinear, false);

		d_Rotate2DFT(t_Re, t_Im, d_output + ElementsFFT(dims) * b, dims, angles[b], maxfreq, mode, isoutputzerocentered);

		hipDestroyTextureObject(t_Re);
		hipDestroyTextureObject(t_Im);
		hipFreeArray(a_Re);
		hipFreeArray(a_Im);
	}

	hipFree(d_imag);
	hipFree(d_real);
}

void d_Rotate2DFT(cudaTex t_inputRe, cudaTex t_inputIm, tcomplex* d_output, int3 dims, tfloat angle, tfloat maxfreq, T_INTERP_MODE mode, bool isoutputzerocentered)
{
		glm::mat2 rotation = Matrix2Rotation(-angle);

		dim3 TpB = dim3(16, 16);
		dim3 grid = dim3((dims.x / 2 + 1 + 15) / 16, (dims.y + 15) / 16);

		if (isoutputzerocentered)
		{
			if (mode == T_INTERP_LINEAR)
				Rotate2DFTKernel<false, true> << <grid, TpB >> > (t_inputRe, t_inputIm, d_output, dims, rotation, maxfreq);
			else if (mode == T_INTERP_CUBIC)
				Rotate2DFTKernel<true, true> << <grid, TpB >> > (t_inputRe, t_inputIm, d_output, dims, rotation, maxfreq);
		}
		else
		{
			if (mode == T_INTERP_LINEAR)
				Rotate2DFTKernel<false, false> << <grid, TpB >> > (t_inputRe, t_inputIm, d_output, dims, rotation, maxfreq);
			else if (mode == T_INTERP_CUBIC)
				Rotate2DFTKernel<true, false> << <grid, TpB >> > (t_inputRe, t_inputIm, d_output, dims, rotation, maxfreq);
		}
}

void d_Rotate2D(tfloat* d_input, tfloat* d_output, int3 dims, tfloat* angles, int batch)
{
	int3 dimspadded = toInt3(dims.x * 2, dims.y * 2, 1);
	tcomplex* d_padded;
	hipMalloc((void**)&d_padded, ElementsFFT(dimspadded) * batch * sizeof(tcomplex));

	d_Pad(d_input, (tfloat*)d_padded, dims, dimspadded, T_PAD_VALUE, (tfloat)0, batch);
	d_RemapFull2FullFFT((tfloat*)d_padded, (tfloat*)d_padded, dimspadded, batch);
	d_FFTR2C((tfloat*)d_padded, d_padded, 2, dimspadded, batch);
	d_RemapHalfFFT2Half(d_padded, d_padded, dimspadded, batch);

	d_Rotate2DFT(d_padded, d_padded, dimspadded, angles, dimspadded.x / 2, T_INTERP_CUBIC, false, batch);

	//d_RemapHalf2HalfFFT(d_padded, d_padded, dimspadded, batch);
	d_IFFTC2R(d_padded, (tfloat*)d_padded, 2, dimspadded, batch);
	d_RemapFullFFT2Full((tfloat*)d_padded, (tfloat*)d_padded, dimspadded, batch);
	d_Pad((tfloat*)d_padded, d_output, dimspadded, dims, T_PAD_VALUE, (tfloat)0, batch);

	hipFree(d_padded);
}


//////////////////////////////
//Rotate 3D in Fourier space//
//////////////////////////////

void d_Rotate3DFT(tcomplex* d_volume, tcomplex* d_output, int3 dims, tfloat3* h_angles, int nangles, T_INTERP_MODE mode, bool iszerocentered)
{
	int3 dimsfft = toInt3(dims.x / 2 + 1, dims.y, dims.z);
	tfloat* d_tempRe;
	hipMalloc((void**)&d_tempRe, ElementsFFT(dims) * sizeof(tfloat));
	tfloat* d_tempIm;
	hipMalloc((void**)&d_tempIm, ElementsFFT(dims) * sizeof(tfloat));

	hipArray* a_Re, *a_Im;
	cudaTex t_Re, t_Im;

	d_ConvertTComplexToSplitComplex(d_volume, d_tempRe, d_tempIm, ElementsFFT(dims));
	if (mode == T_INTERP_CUBIC)
	{
		d_CubicBSplinePrefilter3D(d_tempRe, dimsfft);
		d_CubicBSplinePrefilter3D(d_tempIm, dimsfft);
	}
	d_BindTextureTo3DArray(d_tempRe, a_Re, t_Re, dimsfft, hipFilterModeLinear, false);
	d_BindTextureTo3DArray(d_tempIm, a_Im, t_Im, dimsfft, hipFilterModeLinear, false);
	hipFree(d_tempRe);
	hipFree(d_tempIm);

	d_Rotate3DFT(t_Re, t_Im, d_output, dims, h_angles, nangles, mode, iszerocentered);

	hipDestroyTextureObject(t_Re);
	hipDestroyTextureObject(t_Im);
	hipFreeArray(a_Re);
	hipFreeArray(a_Im);
}

void d_Rotate3DFT(cudaTex t_Re, cudaTex t_Im, tcomplex* d_output, int3 dims, tfloat3* h_angles, int nangles, T_INTERP_MODE mode, bool iszerocentered)
{
	glm::mat4* h_transform = (glm::mat4*)malloc(nangles * sizeof(glm::mat4));
	for (int b = 0; b < nangles; b++)
		h_transform[b] = glm::transpose(Matrix4Euler(h_angles[b])) *
							  Matrix4Translation(tfloat3(-dims.x / 2, -dims.y / 2, -dims.z / 2));
	glm::mat4* d_transform = (glm::mat4*)CudaMallocFromHostArray(h_transform, nangles * sizeof(glm::mat4));

	float maxfreq2 = (float)(dims.x * dims.x / 4);

	dim3 TpB = dim3(16, 16);
	dim3 grid = dim3((dims.x / 2 + 1 + 15) / 16, (dims.y + 15) / 16, dims.z * nangles);
	if (iszerocentered)
	{
		if (mode == T_INTERP_LINEAR)
			Rotate3DFTKernel<false, true> << <grid, TpB >> > (t_Re, t_Im, d_output, dims, d_transform, maxfreq2);
		if (mode == T_INTERP_CUBIC)
			Rotate3DFTKernel<true, true> << <grid, TpB >> > (t_Re, t_Im, d_output, dims, d_transform, maxfreq2);
	}
	else
	{
		if (mode == T_INTERP_LINEAR)
			Rotate3DFTKernel<false, false> << <grid, TpB >> > (t_Re, t_Im, d_output, dims, d_transform, maxfreq2);
		if (mode == T_INTERP_CUBIC)
			Rotate3DFTKernel<true, false> << <grid, TpB >> > (t_Re, t_Im, d_output, dims, d_transform, maxfreq2);
	}

	hipFree(d_transform);
	free(h_transform);
}

void d_Rotate3DFT(tfloat* d_volume, tfloat* d_output, int3 dims, tfloat3* h_angles, int nangles, T_INTERP_MODE mode, bool iszerocentered)
{
	int3 dimsfft = toInt3(dims.x / 2 + 1, dims.y, dims.z);
	tfloat* d_tempRe;
	hipMalloc((void**)&d_tempRe, ElementsFFT(dims) * sizeof(tfloat));

	hipArray* a_Re;
	cudaTex t_Re;

	hipMemcpy(d_tempRe, d_volume, ElementsFFT(dims) * sizeof(tfloat), hipMemcpyDeviceToDevice);
	if (mode == T_INTERP_CUBIC)
		d_CubicBSplinePrefilter3D(d_tempRe, dimsfft);
	d_BindTextureTo3DArray(d_tempRe, a_Re, t_Re, dimsfft, hipFilterModeLinear, false);
	hipFree(d_tempRe);

	d_Rotate3DFT(t_Re, d_output, dims, h_angles, nangles, mode, iszerocentered);

	hipDestroyTextureObject(t_Re);
	hipFreeArray(a_Re);
}

void d_Rotate3DFT(cudaTex t_volume, tfloat* d_output, int3 dims, tfloat3* h_angles, int nangles, T_INTERP_MODE mode, bool iszerocentered)
{
	glm::mat4* h_transform = (glm::mat4*)malloc(nangles * sizeof(glm::mat4));
	for (int b = 0; b < nangles; b++)
		h_transform[b] = glm::transpose(Matrix4Euler(h_angles[b])) *
						 Matrix4Translation(tfloat3(-dims.x / 2, -dims.y / 2, -dims.z / 2));
	glm::mat4* d_transform = (glm::mat4*)CudaMallocFromHostArray(h_transform, nangles * sizeof(glm::mat4));

	float maxfreq2 = (float)(dims.x * dims.x / 4);

	dim3 TpB = dim3(16, 16);
	dim3 grid = dim3((dims.x / 2 + 1 + 15) / 16, (dims.y + 15) / 16, dims.z * nangles);
	if (iszerocentered)
	{
		if (mode == T_INTERP_LINEAR)
			Rotate3DFTKernel<false, true> << <grid, TpB >> > (t_volume, d_output, dims, d_transform, maxfreq2);
		if (mode == T_INTERP_CUBIC)
			Rotate3DFTKernel<true, true> << <grid, TpB >> > (t_volume, d_output, dims, d_transform, maxfreq2);
	}
	else
	{
		if (mode == T_INTERP_LINEAR)
			Rotate3DFTKernel<false, false> << <grid, TpB >> > (t_volume, d_output, dims, d_transform, maxfreq2);
		if (mode == T_INTERP_CUBIC)
			Rotate3DFTKernel<true, false> << <grid, TpB >> > (t_volume, d_output, dims, d_transform, maxfreq2);
	}

	hipFree(d_transform);
	free(h_transform);
}


////////////////
//CUDA kernels//
////////////////

template<bool cubicinterp, bool zerocentered> __global__ void Rotate3DKernel(cudaTex t_input, tfloat* d_output, int3 dims, glm::mat4 transform)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx >= dims.x)
		return;
	int idy = blockIdx.y * blockDim.y + threadIdx.y;
	if (idy >= dims.y)
		return;
	int idz = blockIdx.z;

	int x, y, z;
	if (zerocentered)
	{
		x = idx;
		y = idy;
		z = idz;
	}
	else
	{
		x = dims.x / 2 - idx;
		y = dims.y - 1 - ((idy + dims.y / 2 - 1) % dims.y);
		z = dims.z - 1 - ((idz + dims.z / 2 - 1) % dims.z);
	}

	glm::vec4 pos = transform * glm::vec4(x, y, z, 1);
	tfloat value;
	if(pos.x >= 0.0f && pos.x < (float)dims.x && pos.y >= 0.0f && pos.y < (float)dims.y && pos.z >= 0.0f && pos.z < (float)dims.z)
	{
		if (cubicinterp)
			value = cubicTex3DSimple<tfloat>(t_input, pos.x, pos.y, pos.z);
		else
			value = tex3D<tfloat>(t_input, pos.x, pos.y, pos.z);
	}
	else
		value = (tfloat)0;

	d_output[(idz * dims.y + idy) * dims.x + idx] = value;
}

template<bool cubicinterp, bool outputzerocentered> __global__ void Rotate2DFTKernel(cudaTex t_Re, cudaTex t_Im, tcomplex* d_output, int3 dims, glm::mat2 transform, tfloat maxfreq)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if(idx > dims.x / 2)
		return;
	int idy = blockIdx.y * blockDim.y + threadIdx.y;
	if (idy >= dims.y)
		return;

	int x, y;
	if (outputzerocentered)
	{
		x = idx;
		y = idy;
	}
	else
	{
		x = dims.x / 2 - idx;
		y = dims.y - 1 - ((idy + dims.y / 2 - 1) % dims.y);
	}

	glm::vec2 pos = transform * glm::vec2(idx - dims.x / 2, idy - dims.y / 2);

	if (glm::length(pos) > maxfreq)
	{
		d_output[y * (dims.x / 2 + 1) + x] = make_hipComplex(0.0f, 0.0f);
		return;
	}

	bool isnegative = false;
	if(pos.x > 0.00001f)
	{
		pos = -pos;
		isnegative = true;
	}

	pos += glm::vec2((float)(dims.x / 2) + 0.5f, (float)(dims.y / 2) + 0.5f);
	
	tfloat valre, valim;
	if (!cubicinterp)
	{
		valre = tex2D<tfloat>(t_Re, pos.x, pos.y);
		valim = tex2D<tfloat>(t_Im, pos.x, pos.y);
	}
	else
	{
		valre = cubicTex2D(t_Re, pos.x, pos.y);
		valim = cubicTex2D(t_Im, pos.x, pos.y);
	}

	if(isnegative)
		valim = -valim;

	d_output[y * (dims.x / 2 + 1) + x] = make_hipComplex(valre, valim);
}

template<bool cubicinterp, bool zerocentered> __global__ void Rotate3DFTKernel(cudaTex t_Re, cudaTex t_Im, tcomplex* d_output, int3 dims, glm::mat4* d_transform, float maxfreq2)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx > dims.x / 2)
		return;
	uint idglobal = blockIdx.z / dims.z;
	d_output += ElementsFFT(dims) * idglobal;
	d_transform += idglobal;

	int idy = blockIdx.y * blockDim.y + threadIdx.y;
	if (idy >= dims.y)
		return;
	int idz = blockIdx.z % dims.z;

	int x, y, z;
	if (zerocentered)
	{
		x = idx;
		y = idy;
		z = idz;
	}
	else
	{
		x = dims.x / 2 - idx;
		y = dims.y - 1 - ((idy + dims.y / 2 - 1) % dims.y);
		z = dims.z - 1 - ((idz + dims.z / 2 - 1) % dims.z);
	}

	glm::vec4 pos = *d_transform * glm::vec4(x, y, z, 1);

	float radiussq = pos.x * pos.x + pos.y * pos.y + pos.z * pos.z;
	if (radiussq >= maxfreq2)
	{
		d_output[(idz * dims.y + idy) * (dims.x / 2 + 1) + idx] = make_hipComplex(0, 0);
		return;
	}

	bool isnegative = false;
	if (pos.x > 0.0000001f)
	{
		pos = -pos;
		isnegative = true;
	}

	pos += (float)(dims.x / 2) + 0.5f;

	tfloat valre, valim;
	if (!cubicinterp)
	{
		valre = tex3D<tfloat>(t_Re, pos.x, pos.y, pos.z);
		valim = tex3D<tfloat>(t_Im, pos.x, pos.y, pos.z);
	}
	else
	{
		valre = cubicTex3D(t_Re, pos.x, pos.y, pos.z);
		valim = cubicTex3D(t_Im, pos.x, pos.y, pos.z);
	}

	if (isnegative)
		valim = -valim;

	d_output[(idz * dims.y + idy) * (dims.x / 2 + 1) + idx] = make_hipComplex(valre, valim);
}

template<bool cubicinterp, bool zerocentered> __global__ void Rotate3DFTKernel(cudaTex t_Re, tfloat* d_output, int3 dims, glm::mat4* d_transform, float maxfreq2)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx > dims.x / 2)
		return;
	uint idglobal = blockIdx.z / dims.z;
	d_output += ElementsFFT(dims) * idglobal;
	d_transform += idglobal;

	int idy = blockIdx.y * blockDim.y + threadIdx.y;
	if (idy >= dims.y)
		return;
	int idz = blockIdx.z % dims.z;

	int x, y, z;
	if (zerocentered)
	{
		x = idx;
		y = idy;
		z = idz;
	}
	else
	{
		x = dims.x / 2 - idx;
		y = dims.y - 1 - ((idy + dims.y / 2 - 1) % dims.y);
		z = dims.z - 1 - ((idz + dims.z / 2 - 1) % dims.z);
	}

	glm::vec4 pos = *d_transform * glm::vec4(x, y, z, 1);

	float radiussq = pos.x * pos.x + pos.y * pos.y + pos.z * pos.z;
	if (radiussq >= maxfreq2)
	{
		d_output[(idz * dims.y + idy) * (dims.x / 2 + 1) + idx] = 0;
		return;
	}

	if (pos.x > 0.0000001f)
		pos = -pos;

	pos += (float)(dims.x / 2) + 0.5f;

	tfloat valre;
	if (!cubicinterp)
		valre = tex3D<tfloat>(t_Re, pos.x, pos.y, pos.z);
	else
		valre = cubicTex3D(t_Re, pos.x, pos.y, pos.z);

	d_output[(idz * dims.y + idy) * (dims.x / 2 + 1) + idx] = valre;
}