#include "hip/hip_runtime.h"
#include "Prerequisites.cuh"
#include "Helper.cuh"

void d_BindTextureToArray(tfloat* d_input, hipArray* &d_createdarray, hipTextureObject_t &texture, int2 dims, hipTextureFilterMode filtermode, bool normalizedcoords)
{
	hipChannelFormatDesc desc = hipCreateChannelDesc<tfloat>();
	hipArray* a_input;
	hipMallocArray(&a_input, &desc, dims.x, dims.y);
	hipMemcpyToArray(a_input, 0, 0, d_input, dims.x * dims.y * sizeof(tfloat), hipMemcpyDeviceToDevice);

	struct hipResourceDesc resDesc;
	memset(&resDesc, 0, sizeof(resDesc));
	resDesc.resType = hipResourceTypeArray;
	resDesc.res.array.array = a_input;

	struct hipTextureDesc texDesc;
	memset(&texDesc, 0, sizeof(texDesc));
	texDesc.filterMode = filtermode;
	texDesc.readMode = hipReadModeElementType;
	texDesc.normalizedCoords = normalizedcoords;
	texDesc.addressMode[0] = hipAddressModeWrap;
	texDesc.addressMode[1] = hipAddressModeWrap;
	texDesc.addressMode[2] = hipAddressModeWrap;
	hipTextureObject_t texObj = 0;
	hipCreateTextureObject(&texObj, &resDesc, &texDesc, NULL);

	d_createdarray = a_input;
	texture = texObj;
}

void d_BindTextureTo3DArray(tfloat* d_input, hipArray* &d_createdarray, hipTextureObject_t &texture, int3 dims, hipTextureFilterMode filtermode, bool normalizedcoords)
{
	hipChannelFormatDesc desc = hipCreateChannelDesc<tfloat>();
	hipExtent extent = make_hipExtent(dims.x, dims.y, dims.z);
	hipArray* a_input;
	hipMalloc3DArray(&a_input, &desc, extent);

	hipPitchedPtr p_input = CopyVolumeDeviceToDevice(d_input, dims);

	hipMemcpy3DParms p = { 0 };
	p.extent = extent;
	p.srcPtr = p_input;
	p.dstArray = a_input;
	p.kind = hipMemcpyDeviceToDevice;
	hipMemcpy3D(&p);

	hipFree(p_input.ptr);

	struct hipResourceDesc resDesc;
	memset(&resDesc, 0, sizeof(resDesc));
	resDesc.resType = hipResourceTypeArray;
	resDesc.res.array.array = a_input;

	struct hipTextureDesc texDesc;
	memset(&texDesc, 0, sizeof(texDesc));
	texDesc.filterMode = filtermode;
	texDesc.readMode = hipReadModeElementType;
	texDesc.normalizedCoords = normalizedcoords;
	texDesc.addressMode[0] = hipAddressModeWrap;
	texDesc.addressMode[1] = hipAddressModeWrap;
	texDesc.addressMode[2] = hipAddressModeWrap;
	hipTextureObject_t texObj = 0;
	hipCreateTextureObject(&texObj, &resDesc, &texDesc, NULL);

	d_createdarray = a_input;
	texture = texObj;
}