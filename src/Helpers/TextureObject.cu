#include "hip/hip_runtime.h"
#include "Prerequisites.cuh"

void d_BindTextureToArray(tfloat* d_input, hipArray* &d_createdarray, hipTextureObject_t &texture, int2 dims, hipTextureFilterMode filtermode, bool normalizedcoords)
{
	hipChannelFormatDesc desc = hipCreateChannelDesc<tfloat>();
	hipArray* d_inputArray;
	hipMallocArray(&d_inputArray, &desc, dims.x, dims.y);
	hipMemcpyToArray(d_inputArray, 0, 0, d_input, dims.x * dims.y * sizeof(tfloat), hipMemcpyDeviceToDevice);

	struct hipResourceDesc resDesc;
	memset(&resDesc, 0, sizeof(resDesc));
	resDesc.resType = hipResourceTypeArray;
	resDesc.res.array.array = d_inputArray;

	struct hipTextureDesc texDesc;
	memset(&texDesc, 0, sizeof(texDesc));
	texDesc.filterMode = filtermode;
	texDesc.readMode = hipReadModeElementType;
	texDesc.normalizedCoords = normalizedcoords;

	hipTextureObject_t texObj = 0;
	hipCreateTextureObject(&texObj, &resDesc, &texDesc, NULL);

	d_createdarray = d_inputArray;
	texture = texObj;
}