#include "hip/hip_runtime.h"
#include "Prerequisites.cuh"
#include "Angles.cuh"
#include "Binary.cuh"
#include "Correlation.cuh"
#include "FFT.cuh"
#include "Generics.cuh"
#include "Helper.cuh"
#include "ImageManipulation.cuh"
#include "Projection.cuh"
#include "Reconstruction.cuh"
#include "Relion.cuh"
#include "Transformation.cuh"

namespace gtom
{
	__global__ void BatchComplexConjMultiplyKernel(tcomplex* d_input1, tcomplex* d_input2, tcomplex* d_output, uint vectorlength, uint batch);
	__global__ void UpdateCorrelationKernel(tfloat* d_correlation, uint vectorlength, uint batch, int batchoffset, tfloat* d_bestcorrelation, float* d_bestangle);

	void d_PickSubTomograms(tcomplex* d_projectordata,
							tfloat projectoroversample,
							int3 dimsprojector,
							tcomplex* d_experimentalft,
							tfloat* d_ctf,
							int3 dimsvolume,
							uint nvolumes,
							tfloat3* h_angles,
							uint nangles,
							tfloat maskradius,
							tfloat* d_bestcorrelation,
							float* d_bestangle)
	{
		uint batchsize = 64;
		if (nvolumes > batchsize)
			throw;

		d_ValueFill(d_bestcorrelation, Elements(dimsvolume) * nvolumes, (tfloat)-1e30);
		d_ValueFill(d_bestangle, Elements(dimsvolume) * nvolumes, (float)0);

		tfloat3* d_angles = (tfloat3*)CudaMallocFromHostArray(h_angles, nangles * sizeof(tfloat3));

		tcomplex* d_projectedft;
		hipMalloc((void**)&d_projectedft, ElementsFFT(dimsvolume) * batchsize * sizeof(tcomplex));
		tcomplex* d_projectedftctf;
		hipMalloc((void**)&d_projectedftctf, ElementsFFT(dimsvolume) * batchsize * sizeof(tcomplex));
		tcomplex* d_projectedftctfcorr;
		hipMalloc((void**)&d_projectedftctfcorr, ElementsFFT(dimsvolume) * batchsize * sizeof(tcomplex));
		tfloat* d_projected;
		hipMalloc((void**)&d_projected, Elements(dimsvolume) * batchsize * sizeof(tfloat));

		hipfftHandle planforw = d_FFTR2CGetPlan(3, dimsvolume, batchsize);
		hipfftHandle planback = d_IFFTC2RGetPlan(3, dimsvolume, batchsize);
		
		for (uint b = 0; b < nangles; b += batchsize)
		{
			uint curbatch = tmin(batchsize, nangles - b);

			d_rlnProject(d_projectordata, dimsprojector, d_projectedft, dimsvolume, h_angles + b, projectoroversample, curbatch);

			// Multiply by experimental CTF, norm in realspace, go back into Fourier space for convolution
			d_ComplexMultiplyByVector(d_projectedft, d_ctf, d_projectedftctf, ElementsFFT(dimsvolume), curbatch);
			d_IFFTC2R(d_projectedftctf, d_projected, &planback);
			d_NormMonolithic(d_projected, d_projected, Elements(dimsvolume), T_NORM_MEAN01STD, curbatch);
			//d_WriteMRC(d_projected, toInt3(dimsvolume.x, dimsvolume.y, dimsvolume.z * curbatch), "d_projected.mrc");
			d_FFTR2C(d_projected, d_projectedftctf, &planforw);

			for (uint v = 0; v < nvolumes; v++)
			{
				// Multiply current experimental volume by conjugate references
				{
					int TpB = 128;
					dim3 grid = dim3((ElementsFFT(dimsvolume) + TpB - 1) / TpB, 1, 1);
					BatchComplexConjMultiplyKernel << <grid, TpB >> > (d_experimentalft + ElementsFFT(dimsvolume) * v, d_projectedftctf, d_projectedftctfcorr, ElementsFFT(dimsvolume), curbatch);
				}

				d_IFFTC2R(d_projectedftctfcorr, d_projected, &planback);
				//d_WriteMRC(d_projected, toInt3(dimsvolume.x, dimsvolume.y, dimsvolume.z * curbatch), "d_correlation_individual.mrc");

				// Update correlation and angles with best values
				{
					int TpB = 128;
					dim3 grid = dim3((ElementsFFT(dimsvolume) + TpB - 1) / TpB, 1, 1);
					UpdateCorrelationKernel <<<grid, TpB>>> (d_projected, 
															Elements(dimsvolume), 
															curbatch, 
															b,
															d_bestcorrelation + Elements(dimsvolume) * v, 
															d_bestangle + Elements(dimsvolume) * v);
				}
				
				//d_WriteMRC(d_bestcorrelation + Elements(dimsvolume) * v, dimsvolume, "d_correlation_best.mrc");
			}
		}

		hipfftDestroy(planforw);
		hipfftDestroy(planback);

		// Normalize correlation by local standard deviation
		{
			d_IFFTC2R(d_experimentalft, d_projected, 3, dimsvolume, nvolumes, false);
			hipfftHandle planforwstd = d_FFTR2CGetPlan(3, dimsvolume);
			hipfftHandle planbackstd = d_IFFTC2RGetPlan(3, dimsvolume);

			for (uint v = 0; v < nvolumes; v++)
				d_LocalStd(d_projected + Elements(dimsvolume) * v, dimsvolume, maskradius, d_projected + Elements(dimsvolume) * v, NULL, planforwstd, planbackstd);

			hipfftDestroy(planbackstd);
			hipfftDestroy(planforwstd);

			//d_WriteMRC(d_projected, toInt3(dimsvolume.x, dimsvolume.y, dimsvolume.z * nvolumes), "d_localstd.mrc");

			d_DivideSafeByVector(d_bestcorrelation, d_projected, d_bestcorrelation, Elements(dimsvolume) * nvolumes);
		}

		hipFree(d_projected);
		hipFree(d_projectedftctfcorr);
		hipFree(d_projectedftctf);
		hipFree(d_projectedft);
		hipFree(d_angles);
	}

	__global__ void BatchComplexConjMultiplyKernel(tcomplex* d_input1, tcomplex* d_input2, tcomplex* d_output, uint vectorlength, uint batch)
	{
		for (uint id = blockIdx.x * blockDim.x + threadIdx.x; id < vectorlength; id += gridDim.x * blockDim.x)
		{
			tcomplex input1 = d_input1[id];

			for (uint b = 0; b < batch; b++)
				d_output[b * vectorlength + id] = cmul(input1, cconj(d_input2[b * vectorlength + id]));
		}
	}

	__global__ void UpdateCorrelationKernel(tfloat* d_correlation, uint vectorlength, uint batch, int batchoffset, tfloat* d_bestcorrelation, float* d_bestangle)
	{
		for (uint id = blockIdx.x * blockDim.x + threadIdx.x; id < vectorlength; id += gridDim.x * blockDim.x)
		{
			tfloat bestcorrelation = d_bestcorrelation[id];
			float bestangle = d_bestangle[id];

			for (uint b = 0; b < batch; b++)
			{
				tfloat newcorrelation = d_correlation[b * vectorlength + id];
				if (newcorrelation > bestcorrelation)
				{
					bestcorrelation = newcorrelation;
					bestangle = b + batchoffset;
				}
			}

			d_bestcorrelation[id] = bestcorrelation;
			d_bestangle[id] = bestangle;
		}
	}
}