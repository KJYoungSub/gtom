#include "hip/hip_runtime.h"
#include "Prerequisites.cuh"


////////////////////////////
//CUDA kernel declarations//
////////////////////////////

template<class T> __global__ void ReduceAddKernel(T* d_input, T* d_output, int nvectors, int vectorlength);
template<class T> __global__ void ReduceMeanKernel(T* d_input, T* d_output, int nvectors, int vectorlength);
template<class T> __global__ void ReduceMeanWeightedKernel(T* d_input, tfloat* d_inputweights, T* d_output, int nvectors, int vectorlength);


////////////
//Addition//
////////////

template<class T> void d_ReduceAdd(T* d_input, T* d_output, int vectorlength, int nvectors, int batch)
{
	int TpB = min(NextMultipleOf(nvectors, 32), 256);
	dim3 grid = dim3(min(vectorlength, 2048), batch);
	ReduceAddKernel<T> <<<grid, TpB>>> (d_input, d_output, nvectors, vectorlength);
}
template void d_ReduceAdd<char>(char* d_input, char* d_output, int vectorlength, int nvectors, int batch);
template void d_ReduceAdd<short>(short* d_input, short* d_output, int vectorlength, int nvectors, int batch);
template void d_ReduceAdd<int>(int* d_input, int* d_output, int vectorlength, int nvectors, int batch);
template void d_ReduceAdd<uint>(uint* d_input, uint* d_output, int vectorlength, int nvectors, int batch);
template void d_ReduceAdd<float>(float* d_input, float* d_output, int vectorlength, int nvectors, int batch);
template void d_ReduceAdd<double>(double* d_input, double* d_output, int vectorlength, int nvectors, int batch);

template<class T> __global__ void ReduceAddKernel(T* d_input, T* d_output, int nvectors, int vectorlength)
{
	d_input += blockIdx.y * nvectors * vectorlength;

	for (int id = blockIdx.x * blockDim.x + threadIdx.x; id < vectorlength; id += gridDim.x * blockDim.x)
	{
		T sum = (T)0;

		for (int n = 0; n < nvectors; n++)
			sum += d_input[n * vectorlength + id];

		d_output[blockIdx.y * vectorlength + id] = sum;
	}
}


////////
//Mean//
////////

template<class T> void d_ReduceMean(T* d_input, T* d_output, int vectorlength, int nvectors, int batch)
{
	int TpB = min(NextMultipleOf(nvectors, 32), 256);
	dim3 grid = dim3(min(vectorlength, 2048), batch);
	ReduceMeanKernel<T> << <grid, TpB >> > (d_input, d_output, nvectors, vectorlength);
}
template void d_ReduceMean<char>(char* d_input, char* d_output, int vectorlength, int nvectors, int batch);
template void d_ReduceMean<short>(short* d_input, short* d_output, int vectorlength, int nvectors, int batch);
template void d_ReduceMean<int>(int* d_input, int* d_output, int vectorlength, int nvectors, int batch);
template void d_ReduceMean<uint>(uint* d_input, uint* d_output, int vectorlength, int nvectors, int batch);
template void d_ReduceMean<float>(float* d_input, float* d_output, int vectorlength, int nvectors, int batch);
template void d_ReduceMean<double>(double* d_input, double* d_output, int vectorlength, int nvectors, int batch);

template<class T> __global__ void ReduceMeanKernel(T* d_input, T* d_output, int nvectors, int vectorlength)
{
	d_input += blockIdx.y * nvectors * vectorlength;

	for (int id = blockIdx.x * blockDim.x + threadIdx.x; id < vectorlength; id += gridDim.x * blockDim.x)
	{
		T sum = (T)0;

		for (int n = 0; n < nvectors; n++)
			sum += d_input[n * vectorlength + id];

		d_output[blockIdx.y * vectorlength + id] = sum / (T)nvectors;
	}
}


/////////////////
//Mean weighted//
/////////////////

template<class T> void d_ReduceMeanWeighted(T* d_input, tfloat* d_inputweights, T* d_output, int vectorlength, int nvectors, int batch)
{
	int TpB = min(NextMultipleOf(nvectors, 32), 256);
	dim3 grid = dim3(min(vectorlength, 2048), batch);
	ReduceMeanWeightedKernel<T> << <grid, TpB >> > (d_input, d_inputweights, d_output, nvectors, vectorlength);
}
template void d_ReduceMeanWeighted<char>(char* d_input, tfloat* d_inputweights, char* d_output, int vectorlength, int nvectors, int batch);
template void d_ReduceMeanWeighted<short>(short* d_input, tfloat* d_inputweights, short* d_output, int vectorlength, int nvectors, int batch);
template void d_ReduceMeanWeighted<int>(int* d_input, tfloat* d_inputweights, int* d_output, int vectorlength, int nvectors, int batch);
template void d_ReduceMeanWeighted<uint>(uint* d_input, tfloat* d_inputweights, uint* d_output, int vectorlength, int nvectors, int batch);
template void d_ReduceMeanWeighted<float>(float* d_input, tfloat* d_inputweights, float* d_output, int vectorlength, int nvectors, int batch);
template void d_ReduceMeanWeighted<double>(double* d_input, tfloat* d_inputweights, double* d_output, int vectorlength, int nvectors, int batch);

template<class T> __global__ void ReduceMeanWeightedKernel(T* d_input, tfloat* d_inputweights, T* d_output, int nvectors, int vectorlength)
{
	d_input += blockIdx.y * nvectors * vectorlength;

	for (int id = blockIdx.x * blockDim.x + threadIdx.x; id < vectorlength; id += gridDim.x * blockDim.x)
	{
		T sum = (T)0;
		tfloat weightsum = 0;

		for (int n = 0; n < nvectors; n++)
		{
			tfloat weight = d_inputweights[n * vectorlength + id];
			weightsum += weight;
			sum += d_input[n * vectorlength + id] * weight;
		}

		d_output[blockIdx.y * vectorlength + id] = sum / max((tfloat)1, weightsum);
	}
}