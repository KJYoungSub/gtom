#include "hip/hip_runtime.h"
#include "Prerequisites.cuh"
#include "FFT.cuh"
#include "Generics.cuh"
#include "Helper.cuh"
#include "Masking.cuh"


namespace gtom
{
	void d_LocalStd(tfloat* d_map, int3 dimsmap, tfloat localradius, tfloat* d_std, tfloat* d_mean)
	{
		tcomplex* d_maskft = CudaMallocValueFilled(ElementsFFT(dimsmap), make_hipComplex(1, 1));
		tfloat masksum = 0;

		// Create spherical mask, calculate its sum, and pre-FFT it for convolution
		{
			d_SphereMask((tfloat*)d_maskft, (tfloat*)d_maskft, dimsmap, &localradius, 0, NULL);
			d_RemapFull2FullFFT((tfloat*)d_maskft, (tfloat*)d_maskft, dimsmap);

			tfloat* d_sum = CudaMallocValueFilled(1, (tfloat)0);
			d_Sum((tfloat*)d_maskft, d_sum, Elements(dimsmap));
			hipMemcpy(&masksum, d_sum, sizeof(tfloat), hipMemcpyDeviceToHost);
			hipFree(d_sum);

			d_FFTR2C((tfloat*)d_maskft, d_maskft, DimensionCount(dimsmap), dimsmap);
		}

		tcomplex* d_mapft;
		hipMalloc((void**)&d_mapft, ElementsFFT(dimsmap) * sizeof(tcomplex));
		tcomplex* d_map2ft;
		hipMalloc((void**)&d_map2ft, ElementsFFT(dimsmap) * sizeof(tcomplex));
		
		// Create FTs of map and map^2
		{
			d_FFTR2C(d_map, d_mapft, DimensionCount(dimsmap), dimsmap);

			d_Square(d_map, (tfloat*)d_map2ft, Elements(dimsmap));
			d_FFTR2C((tfloat*)d_map2ft, d_map2ft, DimensionCount(dimsmap), dimsmap);
		}

		// Convolute
		{
			d_ComplexMultiplyByConjVector(d_mapft, d_maskft, d_mapft, ElementsFFT(dimsmap));
			d_ComplexMultiplyByConjVector(d_map2ft, d_maskft, d_map2ft, ElementsFFT(dimsmap));

			d_IFFTC2R(d_mapft, (tfloat*)d_mapft, DimensionCount(dimsmap), dimsmap);
			d_IFFTC2R(d_map2ft, (tfloat*)d_map2ft, DimensionCount(dimsmap), dimsmap);
		}

		// Optionally, also output local mean
		if (d_mean != NULL)
		{
			d_DivideByScalar((tfloat*)d_mapft, d_mean, Elements(dimsmap), masksum);
		}

		// std = sqrt(max(0, masksum * conv2 - conv1^2)) / masksum
		{
			d_MultiplyByScalar((tfloat*)d_map2ft, (tfloat*)d_map2ft, Elements(dimsmap), masksum);
			d_Square((tfloat*)d_mapft, (tfloat*)d_mapft, Elements(dimsmap));

			d_SubtractVector((tfloat*)d_map2ft, (tfloat*)d_mapft, (tfloat*)d_map2ft, Elements(dimsmap));
			d_MaxOp((tfloat*)d_map2ft, (tfloat)0, (tfloat*)d_map2ft, Elements(dimsmap));

			d_Sqrt((tfloat*)d_map2ft, (tfloat*)d_map2ft, Elements(dimsmap));

			d_DivideByScalar((tfloat*)d_map2ft, d_std, Elements(dimsmap), masksum);
		}

		hipFree(d_map2ft);
		hipFree(d_mapft);
		hipFree(d_maskft);
	}
}